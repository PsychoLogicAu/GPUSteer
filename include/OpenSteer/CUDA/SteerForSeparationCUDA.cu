#include "hip/hip_runtime.h"
#include "SteerForSeparationCUDA.cuh"

using namespace OpenSteer;

extern "C"
{
	__global__ void SteerForSeparationKernel(	uint const*		pdKNNIndices,
												size_t const	k,
												
												float3 const*	pdPosition,
		
												float3 *		pdSteering,
												size_t const	numAgents,
												float const		fWeight,

												uint *			pdAppliedKernels,
												uint const		doNotApplyWith
												);
}

SteerForSeparationCUDA::SteerForSeparationCUDA(	AgentGroup * pAgentGroup, KNNData * pKNNData, AgentGroup * pOtherGroup, float const fWeight, uint const doNotApplyWith )
:	AbstractCUDAKernel( pAgentGroup, fWeight, doNotApplyWith ),
	m_pKNNData( pKNNData ),
	m_pOtherGroup( pOtherGroup )
{
	// Nothing to do.
}

void SteerForSeparationCUDA::init( void )
{
	// Nothing to do.
}

void SteerForSeparationCUDA::run( void )
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	size_t const	numAgents			= getNumAgents();
	size_t const	k					= m_pKNNData->k();

	// Gather required device pointers.
	float3 const*	pdPosition			= m_pAgentGroupData->pdPosition();
	float3 *		pdSteering			= m_pAgentGroupData->pdSteering();
	uint const*		pdKNNIndices		= m_pKNNData->pdKNNIndices();

	uint *			pdAppliedKernels	= m_pAgentGroupData->pdAppliedKernels();

	// Compute the size of shared memory needed for each block.
	size_t shMemSize = k * THREADSPERBLOCK * sizeof(uint);

	// Launch the kernel.
	SteerForSeparationKernel<<< grid, block, shMemSize >>>( pdKNNIndices, k, pdPosition, pdSteering, numAgents, m_fWeight, pdAppliedKernels, m_doNotApplyWith );
	cutilCheckMsg( "SteerForSeparationKernel failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

void SteerForSeparationCUDA::close( void )
{
	// The AgentGroup data has most likely changed.
	m_pAgentGroup->SetSyncHost();
}
