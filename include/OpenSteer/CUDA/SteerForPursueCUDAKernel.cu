#include "hip/hip_runtime.h"
#include "../AgentGroupData.cuh"
#include "../VectorUtils.cuh"

#include "CUDAKernelGlobals.cuh"

using namespace OpenSteer;

extern "C"
{
__global__ void SteerForPursueCUDAKernel(	float4 const* pdPosition,
											float4 const* pdDirection,
											float const* pdSpeed, 

											float3 const targetPosition,
											float3 const targetForward,
											float3 const targetVelocity,
											float const targetSpeed,

											float4 * pdSteering,

											size_t const numAgents,
											float const maxPredictionTime,
											float const fWeight,
											uint * pdAppliedKernels,
											uint const doNotApplyWith
											);
}

__global__ void SteerForPursueCUDAKernel(	float4 const* pdPosition,
											float4 const* pdDirection,
											float const* pdSpeed, 

											float3 const targetPosition,
											float3 const targetForward,
											float3 const targetVelocity,
											float const targetSpeed,

											float4 * pdSteering,

											size_t const numAgents,
											float const maxPredictionTime,
											float const fWeight,
											uint * pdAppliedKernels,
											uint const doNotApplyWith
											)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( index >= numAgents )
		return;

	if( pdAppliedKernels[ index ] & doNotApplyWith )
		return;

	// Declare shared memory.
	__shared__ float3 shPosition[THREADSPERBLOCK];
	__shared__ float3 shDirection[THREADSPERBLOCK];
	__shared__ float3 shSteering[THREADSPERBLOCK];
	__shared__ float shSpeed[THREADSPERBLOCK];

	POSITION_SH( threadIdx.x ) = POSITION_F3( index );
	DIRECTION_SH( threadIdx.x ) = DIRECTION_F3( index );
	STEERING_SH( threadIdx.x ) = STEERING_F3( index );
	SPEED_SH( threadIdx.x ) = SPEED( index );
	__syncthreads();

	float3 steering = { 0.f, 0.f, 0.f };

	// If the target is ahead, just seek to its current position.
	float3 const toTarget = float3_subtract( targetPosition, POSITION_SH( threadIdx.x ) );
	float const relativeHeading = float3_dot( DIRECTION_SH( threadIdx.x ), targetForward );

	if( (relativeHeading < -0.95f) && float3_dot( toTarget, DIRECTION_SH( threadIdx.x ) ) > 0 )
	{
		// Get the desired velocity.
		float3 const desiredVelocity = float3_subtract( targetPosition, POSITION_SH( threadIdx.x ) );

		// Set the steering vector.
		steering = float3_subtract( desiredVelocity, DIRECTION_SH( threadIdx.x ) );
	}
	else
	{
		float lookAheadTime = float3_length( toTarget ) / ( SPEED_SH( threadIdx.x ) + targetSpeed );
		float3 newTarget = float3_add( targetPosition, float3_scalar_multiply( targetVelocity, (maxPredictionTime < lookAheadTime) ? maxPredictionTime : lookAheadTime ) );

		// Get the desired velocity.
		float3 desiredVelocity = float3_subtract( newTarget, POSITION_SH( threadIdx.x ) );

		// Set the steering vector.
		steering = float3_subtract( desiredVelocity, DIRECTION_SH( threadIdx.x ) );
	}

	// Normalize and apply the weight.
	steering = float3_scalar_multiply( float3_normalize( steering ), fWeight );

	// Set the applied kernel bit.
	if( ! float3_equals( steering, float3_zero() ) )
		pdAppliedKernels[ index ] |= KERNEL_PURSUE_BIT;

	// Add into the steering vector.
	STEERING_SH( threadIdx.x ) = float3_add( steering, STEERING_SH( threadIdx.x ) );

	// Write to global memory.
	STEERING( index ) = STEERING_SH_F4( threadIdx.x );
}
