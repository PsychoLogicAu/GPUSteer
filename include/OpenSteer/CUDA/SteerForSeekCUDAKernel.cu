#include "hip/hip_runtime.h"
#include "SteerForSeekCUDA.h"

#include "../AgentGroupData.cuh"
#include "../VectorUtils.cuh"

#include "CUDAKernelGlobals.cuh"

using namespace OpenSteer;

extern "C"
{
	__global__ void SteerForSeekCUDAKernel( float3 * pdSteering, float3 const* pdPosition, float3 const* pdForward, float3 const target, size_t const numAgents, float const fWeight );
}

__global__ void SteerForSeekCUDAKernel( float3 * pdSteering, float3 const* pdPosition, float3 const* pdForward, float3 const target, size_t const numAgents, float const fWeight )
{
	int offset = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( offset >= numAgents )
		return;

	__shared__ float3 shSteering[THREADSPERBLOCK];
	__shared__ float3 shPosition[THREADSPERBLOCK];
	__shared__ float3 shForward[THREADSPERBLOCK];

	FLOAT3_GLOBAL_READ( shSteering, pdSteering );
	FLOAT3_GLOBAL_READ( shPosition, pdPosition );
	FLOAT3_GLOBAL_READ( shForward, pdForward );

	// If we already have a steering vector set, do nothing.
	if( ! float3_equals( STEERING_SH( threadIdx.x ), float3_zero() ) )
		return;

	// Get the desired velocity.
	float3 const desiredVelocity = float3_subtract( target, POSITION_SH( threadIdx.x ) );

	// Set the steering vector.
	float3 steering = float3_subtract( desiredVelocity, FORWARD_SH( threadIdx.x ) );

	// Normalize and apply the weight.
	steering = float3_scalar_multiply( float3_normalize( steering ), fWeight );

	// Add into the steering vector.
	STEERING_SH( threadIdx.x ) = float3_add( steering, STEERING_SH( threadIdx.x ) );

	// Copy the steering vectors back to global memory.
	FLOAT3_GLOBAL_WRITE( pdSteering, shSteering );
}
