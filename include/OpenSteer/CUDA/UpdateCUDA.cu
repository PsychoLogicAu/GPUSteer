#include "hip/hip_runtime.h"
#include "UpdateCUDA.h"

using namespace OpenSteer;

// Kernel function prototype.
extern "C"
{
	__global__ void UpdateCUDAKernel(	// vehicle_group_data members.
										float3 * pdSide, float3 * pdUp, float3 * pdForward,
										float3 * pdPosition, float3 * pdSteering, float * pdSpeed,
										// vehicle_group_const members.
										float const* pdMaxForce, float const* pdMaxSpeed, float const* pdMass,
										float const elapsedTime, size_t const numAgents );
}

UpdateCUDA::UpdateCUDA( VehicleGroup * pVehicleGroup, const float fElapsedTime )
:	AbstractCUDAKernel( pVehicleGroup ),
	m_fElapsedTime( fElapsedTime )
{
}

void UpdateCUDA::init( void )
{
	// Nothing to do.
}

void UpdateCUDA::run(void)
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	// Gather pointers to the required data...
	float3 * pdSide = m_pVehicleGroupData->pdSide();
	float3 * pdUp = m_pVehicleGroupData->pdUp();
	float3 * pdForward = m_pVehicleGroupData->pdForward();
	float3 * pdPosition = m_pVehicleGroupData->pdPosition();
	float3 * pdSteering = m_pVehicleGroupData->pdSteering();
	float * pdSpeed = m_pVehicleGroupData->pdSpeed();

	float const* pdMaxForce = m_pVehicleGroupConst->pdMaxForce();
	float const* pdMaxSpeed = m_pVehicleGroupConst->pdMaxSpeed();
	float const* pdMass = m_pVehicleGroupConst->pdMass();

	UpdateCUDAKernel<<< grid, block >>>(	pdSide, pdUp, pdForward, pdPosition, pdSteering, pdSpeed,
											pdMaxForce, pdMaxSpeed, pdMass,
											m_fElapsedTime, getNumAgents() );

	hipDeviceSynchronize();
}

void UpdateCUDA::close(void)
{
	// Device data has changed. Instruct the VehicleGroup it needs to synchronize the host.
	m_pVehicleGroup->SetSyncHost();
}
