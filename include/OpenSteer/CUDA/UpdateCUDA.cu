#include "hip/hip_runtime.h"
#include "UpdateCUDA.h"

using namespace OpenSteer;

// Kernel function prototype.
extern "C"
{
	__global__ void UpdateCUDAKernel(	// vehicle_group_data members.
										float3 * pdSide, float3 * pdUp, float3 * pdForward,
										float3 * pdPosition, float3 * pdSteering, float * pdSpeed,
										// vehicle_group_const members.
										float const* pdMaxForce, float const* pdMaxSpeed, float const* pdMass,
										float const elapsedTime, size_t const numAgents );
}

UpdateCUDA::UpdateCUDA( VehicleGroup * pVehicleGroup, const float fElapsedTime )
:	AbstractCUDAKernel( pVehicleGroup ),
	m_fElapsedTime( fElapsedTime )
{
}

void UpdateCUDA::init( void )
{
	// Nothing to do.
}

void UpdateCUDA::run(void)
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	// Gather pointers to the required data...
	float3 * pdSide = m_pdVehicleGroupData->pdSide();
	float3 * pdUp = m_pdVehicleGroupData->pdUp();
	float3 * pdForward = m_pdVehicleGroupData->pdForward();
	float3 * pdPosition = m_pdVehicleGroupData->pdPosition();
	float3 * pdSteering = m_pdVehicleGroupData->pdSteering();
	float * pdSpeed = m_pdVehicleGroupData->pdSpeed();

	float const* pdMaxForce = m_pdVehicleGroupConst->pdMaxForce();
	float const* pdMaxSpeed = m_pdVehicleGroupConst->pdMaxSpeed();
	float const* pdMass = m_pdVehicleGroupConst->pdMass();

	UpdateCUDAKernel<<< grid, block >>>(	pdSide, pdUp, pdForward, pdPosition, pdSteering, pdSpeed,
											pdMaxForce, pdMaxSpeed, pdMass,
											m_fElapsedTime, getNumAgents() );

	hipDeviceSynchronize();
}

void UpdateCUDA::close(void)
{
	// Device data has changed. Instruct the VehicleGroup it needs to synchronize the host.
	m_pVehicleGroup->SetSyncHost();
}
