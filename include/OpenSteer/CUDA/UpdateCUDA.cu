#include "hip/hip_runtime.h"
#include "UpdateCUDA.h"

using namespace OpenSteer;

// Kernel function prototype.
extern "C"
{
	__global__ void UpdateCUDAKernel(vehicle_data *vehicleData, vehicle_const *vehicleConst, float elapsedTime, int numAgents);
}

UpdateCUDA::UpdateCUDA(VehicleGroup *pVehicleGroup, const float elapsedTime)
:	AbstractCUDAKernel(pVehicleGroup),
	m_elapsedTime(elapsedTime)
{
	m_threadsPerBlock = THREADSPERBLOCK;
}

void UpdateCUDA::init(void)
{
	// Allocate device memory.
	HANDLE_ERROR(hipMalloc((void**)&m_pdVehicleData, getDataSizeInBytes()));
	HANDLE_ERROR(hipMalloc((void**)&m_pdVehicleConst, getConstSizeInBytes()));

	// Copy data to device memory.
	HANDLE_ERROR(hipMemcpy(m_pdVehicleData, (void*)getVehicleData(), getDataSizeInBytes(), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(m_pdVehicleConst, (void*)getVehicleConst(), getConstSizeInBytes(), hipMemcpyHostToDevice));
}

void UpdateCUDA::run(void)
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	UpdateCUDAKernel<<<grid, block>>>(m_pdVehicleData, m_pdVehicleConst, m_elapsedTime, getNumberOfAgents());

	hipDeviceSynchronize();
}

void UpdateCUDA::close(void)
{
	// Copy vehicle data back to the host memory.
	HANDLE_ERROR(hipMemcpy((void*)getVehicleData(), m_pdVehicleData, getDataSizeInBytes(), hipMemcpyDeviceToHost));

	// Deallocate device memory
	HANDLE_ERROR(hipFree(m_pdVehicleData));
	HANDLE_ERROR(hipFree(m_pdVehicleConst));

	m_pdVehicleData = NULL;
	m_pdVehicleConst = NULL;
}
