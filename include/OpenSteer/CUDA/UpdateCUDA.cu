#include "hip/hip_runtime.h"
#include "UpdateCUDA.h"

using namespace OpenSteer;

// Kernel function prototype.
extern "C"
{
	__global__ void UpdateCUDAKernel(	float3 * pdSide,
										float3 * pdUp,
										float4 * pdDirection,
										float4 * pdPosition,

										float4 * pdSteering,
										float * pdSpeed,

										float const* pdMaxForce,
										float const* pdMaxSpeed,
										float const* pdMass,

										float const elapsedTime,
										size_t const numAgents,
										uint * pdAppliedKernels
										);

	// New version.
	__host__ void UpdateCUDAKernelBindTextures(	float4 const*	pdLineStart,
												float4 const*	pdLineEnd,
												float4 const*	pdLineNormal,
												uint const		numLines
												);

	__host__ void UpdateCUDAKernelUnbindTextures( void );

	__global__ void UpdateCUDAKernelNew(		float3 *		pdSide,
												float3 *		pdUp,
												float4 *		pdDirection,
												float4 *		pdPosition,

												float4 *		pdSteering,
												float *			pdSpeed,

												float const*	pdMaxForce,
												float const*	pdMaxSpeed,
												float const*	pdMass,
												float const*	pdRadius,

												uint const*		pdKNLIndices,	// Indices of the K Nearest line segments...
												uint const		k,				// Number of lines in KNL.
												uint const		numLines,

												float const		elapsedTime,
												uint const		numAgents,
												uint *			pdAppliedKernels
												);
}

UpdateCUDA::UpdateCUDA( AgentGroup * pAgentGroup, const float fElapsedTime )
:	AbstractCUDAKernel( pAgentGroup, 1.f, 0 ),
	m_fElapsedTime( fElapsedTime )
{
}

void UpdateCUDA::init( void )
{
	// Nothing to do.
}

void UpdateCUDA::run(void)
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	// Gather pointers to the required data...
	float3 *		pdSide				= m_pAgentGroupData->pdSide();
	float3 *		pdUp				= m_pAgentGroupData->pdUp();
	float4 *		pdDirection			= m_pAgentGroupData->pdDirection();
	float4 *		pdPosition			= m_pAgentGroupData->pdPosition();
	float4 *		pdSteering			= m_pAgentGroupData->pdSteering();
	float *			pdSpeed				= m_pAgentGroupData->pdSpeed();

	float const*	pdMaxForce			= m_pAgentGroupData->pdMaxForce();
	float const*	pdMaxSpeed			= m_pAgentGroupData->pdMaxSpeed();
	float const*	pdMass				= m_pAgentGroupData->pdMass();
	float const*	pdRadius			= m_pAgentGroupData->pdRadius();

	uint *			pdAppliedKernels	= m_pAgentGroupData->pdAppliedKernels();

	uint const&		numAgents			= getNumAgents();

	UpdateCUDAKernel<<< grid, block >>>(	pdSide, pdUp, pdDirection, pdPosition, pdSteering, pdSpeed,
											pdMaxForce, pdMaxSpeed, pdMass,
											m_fElapsedTime, numAgents,
											pdAppliedKernels
											);
	cutilCheckMsg( "UpdateCUDAKernel failed." );
	//CUDA_SAFE_CALL( hipDeviceSynchronize() );

	// Unbind the textures.
	UpdateCUDAKernelUnbindTextures();
}

void UpdateCUDA::close(void)
{
	// Device data has changed. Instruct the AgentGroup it needs to synchronize the host.
	m_pAgentGroup->SetSyncHost();
}

#pragma region UpdateWithAntiPenetrationCUDA

UpdateWithAntiPenetrationCUDA::UpdateWithAntiPenetrationCUDA( AgentGroup * pAgentGroup, KNNData * pKNNData, WallGroup * pWallGroup, const float fElapsedTime )
:	AbstractCUDAKernel( pAgentGroup, 1.f, 0 ),
	m_fElapsedTime( fElapsedTime ),
	m_pKNNData( pKNNData ),
	m_pWallGroup( pWallGroup )
{
}

void UpdateWithAntiPenetrationCUDA::init( void )
{
	// Nothing to do.
}

void UpdateWithAntiPenetrationCUDA::run(void)
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	// Gather pointers to the required data...
	float3 *		pdSide				= m_pAgentGroupData->pdSide();
	float3 *		pdUp				= m_pAgentGroupData->pdUp();
	float4 *		pdDirection			= m_pAgentGroupData->pdDirection();
	float4 *		pdPosition			= m_pAgentGroupData->pdPosition();
	float4 *		pdSteering			= m_pAgentGroupData->pdSteering();
	float *			pdSpeed				= m_pAgentGroupData->pdSpeed();

	float const*	pdMaxForce			= m_pAgentGroupData->pdMaxForce();
	float const*	pdMaxSpeed			= m_pAgentGroupData->pdMaxSpeed();
	float const*	pdMass				= m_pAgentGroupData->pdMass();
	float const*	pdRadius			= m_pAgentGroupData->pdRadius();

	uint *			pdAppliedKernels	= m_pAgentGroupData->pdAppliedKernels();

	uint const&		numAgents			= getNumAgents();

	uint const*		pdKNLIndices		= m_pKNNData->pdKNNIndices();
	uint const&		k					= m_pKNNData->k();

	float4 const*	pdLineStart			= m_pWallGroup->GetWallGroupData().pdLineStart();
	float4 const*	pdLineEnd			= m_pWallGroup->GetWallGroupData().pdLineEnd();
	float4 const*	pdLineNormal		= m_pWallGroup->GetWallGroupData().pdLineNormal();
	uint const&		numLines			= m_pWallGroup->Size();

	size_t const	shMemSize			= k * THREADSPERBLOCK * sizeof(uint);

	// Bind the textures.
	UpdateCUDAKernelBindTextures( pdLineStart, pdLineEnd, pdLineNormal, numLines );

	UpdateCUDAKernelNew<<< grid, block, shMemSize >>>(	pdSide,
														pdUp,
														pdDirection,
														pdPosition,

														pdSteering,
														pdSpeed,
														pdMaxForce,
														pdMaxSpeed,
														pdMass,
														pdRadius,

														pdKNLIndices,
														k,
														numLines,

														m_fElapsedTime,
														numAgents,

														pdAppliedKernels
														);

	// Unbind the textures.
	UpdateCUDAKernelUnbindTextures();
}

void UpdateWithAntiPenetrationCUDA::close(void)
{
	// Device data has changed. Instruct the AgentGroup it needs to synchronize the host.
	m_pAgentGroup->SetSyncHost();
}
#pragma endregion
