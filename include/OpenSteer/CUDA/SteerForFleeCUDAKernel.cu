#include "hip/hip_runtime.h"
#include "SteerForFleeCUDA.h"

#include "../VehicleGroupData.h"
#include "../VectorUtils.cu"

#include "CUDAKernelGlobals.h"

using namespace OpenSteer;

extern "C"
{
	__global__ void SteerForFleeCUDAKernel(	float3 const* pdPosition, float3 const* pdForward, float3 * pdSteering,
											float3 const target, size_t const numAgents )
	{
		int offset = (blockIdx.x * blockDim.x) + threadIdx.x;

		// Check bounds.
		if( offset >= numAgents )
			return;

		// Shared memory for the input data.
		__shared__ float3 shSteering[THREADSPERBLOCK];
		__shared__ float3 shPosition[THREADSPERBLOCK];
		__shared__ float3 shForward[THREADSPERBLOCK];

		// Temporary shared memory storage for desired velocity.
		__shared__ float3 shDesiredVelocity[THREADSPERBLOCK];

		// Copy the required data to shared memory.
		STEERING_SH( threadIdx.x ) = STEERING( offset );
		POSITION_SH( threadIdx.x ) = POSITION( offset );
		FORWARD_SH( threadIdx.x ) = FORWARD( offset );
		
		__syncthreads();

		// If we already have a steering vector set, do nothing.
		if( !float3_equals( STEERING_SH( threadIdx.x ), float3_zero() ) )
			return;

		// Get the desired velocity.
		shDesiredVelocity[ threadIdx.x ] = float3_subtract( POSITION_SH( threadIdx.x ), target );

		// Set the steering vector.
		STEERING_SH( threadIdx.x ) = float3_subtract( shDesiredVelocity[ threadIdx.x ], FORWARD_SH( threadIdx.x ) );

		__syncthreads();

		// Copy the steering vectors back to global memory.
		STEERING( offset ) = STEERING_SH( threadIdx.x );
	}
}