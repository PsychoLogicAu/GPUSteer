#include "hip/hip_runtime.h"
#include "SteerForFleeCUDA.cuh"

#include "../AgentGroupData.cuh"
#include "../VectorUtils.cuh"

#include "CUDAKernelGlobals.cuh"

using namespace OpenSteer;

extern "C"
{
	__global__ void SteerForFleeCUDAKernel(	float3 const*	pdPosition,
											float3 const*	pdForward,
											float3 *		pdSteering,

											float3 const	target,
											size_t const	numAgents,
											float const		fWeight,
											uint *			pdAppliedKernels,
											uint const		doNotApplyWith
											);
}

	__global__ void SteerForFleeCUDAKernel(	float3 const*	pdPosition,
											float3 const*	pdForward,
											float3 *		pdSteering,

											float3 const	target,

											size_t const	numAgents,
											float const		fWeight,
											uint *			pdAppliedKernels,
											uint const		doNotApplyWith
											)

{
	int const index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( index >= numAgents )
		return;

	if( pdAppliedKernels[ index ] & doNotApplyWith )
		return;

	// Shared memory for the input data.
	__shared__ float3 shSteering[THREADSPERBLOCK];
	__shared__ float3 shPosition[THREADSPERBLOCK];
	__shared__ float3 shForward[THREADSPERBLOCK];

	// Copy the required data to shared memory.
	FLOAT3_GLOBAL_READ( shSteering, pdSteering );
	FLOAT3_GLOBAL_READ( shPosition, pdPosition );
	FLOAT3_GLOBAL_READ( shForward, pdForward );

	// Get the desired velocity.
	float3 const desiredVelocity = float3_subtract( POSITION_SH( threadIdx.x ), target );

	// Set the steering vector.
	float3 steering = float3_subtract( desiredVelocity, FORWARD_SH( threadIdx.x ) );

	// Normalize and apply the weight.
	steering = float3_scalar_multiply( float3_normalize( steering ), fWeight );

	// Set the applied kernel bit.
	if( ! float3_equals( steering, float3_zero() ) )
		pdAppliedKernels[ index ] |= KERNEL_FLEE_BIT;

	// Add into the steering vector.
	STEERING_SH( threadIdx.x ) = float3_add( steering, STEERING_SH( threadIdx.x ) );

	// Copy the steering vectors back to global memory.
	FLOAT3_GLOBAL_WRITE( pdSteering, shSteering );
}
