#include "hip/hip_runtime.h"
#include "KNNBinningCUDA.cuh"

#include "../VectorUtils.cuh"

#include "CUDAKernelGlobals.cuh"

using namespace OpenSteer;

// Define the texture reference to access the appropriate bin_cell's index.
texture< uint, hipTextureType3D, hipReadModeElementType > texCellIndices;

// Fetch the bin from texBinCells at a given world {x,y,z} position.
#define CELLINDEX( pos ) ( tex3D( texCellIndices, pos.x, pos.z, pos.y ) )

// Kernel declarations.
extern "C"
{
	// Kernel to set initial bin indices of vehicles in the simulation.
	__global__ void KNNBinningBuildDB(	float3 const*	pdPosition,				// In:	Positions of each vehicle.
										size_t *		pdAgentIndices,			// Out:	Indices of each vehicle.
										size_t *		pdAgentCellIndices,		// Out:	Indices of the bin each vehicle is in.
										size_t const	numAgents				// In:	Number of agents in the simulation.
										);

	// Bind texCellIndices to the hipArray.
	__host__ void KNNBinningCUDABindTexture( hipArray * pCudaArray );
	// Unbind the texture.
	__host__ void KNNBinningCUDAUnbindTexture( void );

	__global__ void KNNBinningKernel(	float3 const*	pdPosition,			// In: Agent positions.
										size_t *		pdAgentIndices,		// In: (sorted) indices of each agent.
										size_t *		pdAgentCellIndices,	// In: (sorted) indices of the cell each agent is in.
										size_t const	k,					// In: Number of neighbors to consider.
										size_t const	radius,				// In: Maximum radius (in cells) to consider.
										size_t const	numAgents,			// In: Number of agents in the simulation.

										uint *			pdKNNIndices,		// Out: indices of K Nearest Neighbors in pdPosition.
										float *			pdKNNDistances,		// Out: distances of the K Nearest Neighbors in pdPosition.
										);
}

//__global__ void KNNBinningKernel(	float3 const*	pdPosition,			// In: Agent positions.
//									uint *			pdKNNIndices,		// Out: indices of K Nearest Neighbors in pdPosition.
//									float *			pdKNNDistances,		// Out: distances of the K Nearest Neighbors in pdPosition.
//									size_t const	k,					// In: Number of neighbors to consider.
//									size_t const	radius,				// In: Maximum radius (in cells) to consider.
//									size_t const	numAgents,			// In: Number of agents in the simulation.
//									)
{
	// Offset of this agent.
	int offset = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( offset > numAgents )
		return;


}

__host__ void KNNBinningCUDABindTexture( hipArray * pdCudaArray )
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uint>();

	texCellIndices.normalized = true;
	texCellIndices.filterMode = hipFilterModePoint;
	texCellIndices.addressMode[0] = hipAddressModeClamp;
	texCellIndices.addressMode[1] = hipAddressModeClamp;
	texCellIndices.addressMode[2] = hipAddressModeClamp;

	CUDA_SAFE_CALL( hipBindTextureToArray( texCellIndices, pdCudaArray, channelDesc ) );
}

__host__ void KNNBinningCUDAUnbindTexture( void )
{
	CUDA_SAFE_CALL( hipUnbindTexture( texCellIndices ) );
}

__global__ void KNNBinningBuildDB(	float3 const*	pdPosition,				// In:	Positions of each vehicle.
									size_t *		pdAgentIndices,			// Out:	Indices of each vehicle.
									size_t *		pdAgentBinIndices,		// Out:	Indices of the bin each vehicle is in.
									size_t const	numAgents				// In:	Number of agents in the simulation.
									)
{
	// Offset of this agent in the global array.
	int offset = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( offset >= numAgents )
		return;

	// Copy the positions to shared memory.
	__shared__ float3 shPosition[THREADSPERBLOCK];
	FLOAT3_COALESCED_READ( shPosition, pdPosition );
	//POSITION_SH( threadIdx.x ) = POSITION( offset );

	// Write the agent's cell index out to global memory.
	pdAgentBinIndices[offset] = CELLINDEX( POSITION_SH( threadIdx.x ) );

	// Write the agent's index out to global memory.
	pdAgentIndices[offset] = offset;
}