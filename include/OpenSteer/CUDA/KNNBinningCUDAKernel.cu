#include "hip/hip_runtime.h"
#include "KNNBinningCUDA.cuh"

#include "../VectorUtils.cuh"

#include "CUDAKernelGlobals.cuh"

using namespace OpenSteer;

// Define the texture reference to access the appropriate bin_cell's index.
texture< uint, hipTextureType3D, hipReadModeElementType > texCellIndicesNormalized;
texture< uint, hipTextureType3D, hipReadModeElementType > texCellIndices;

// Constant memory used to hold the worldSize and worldCells values.
__constant__ float3		constWorldSize;
__constant__ float3		constWorldStep;
__constant__ uint3		constWorldCells;
__constant__ float3		constWorldMin;
__constant__ float3		constWorldMax;

// Fetch the cell index from texCellIndicesNormalized at a given world {x,y,z} position.
#define CELL_INDEX_NORMALIZED( pos )	( tex3D( texCellIndicesNormalized, pos.x, pos.y, pos.z ) )
// Fetch the cell index from texCellIndices at a given texel (x,y,z) coordinate.
#define CELL_INDEX( x, y, z )			( tex3D( texCellIndices, x, y, z ) )

// Kernel declarations.
extern "C"
{
	// Bind the textures to the input hipArray.
	__host__ void KNNBinningCUDABindTexture( hipArray * pCudaArray );
	// Unbind the textures.
	__host__ void KNNBinningCUDAUnbindTexture( void );

	// Kernel to set initial bin indices of vehicles in the simulation.
	__global__ void KNNBinningBuildDB(	float3 const*	pdPosition,				// In:	Positions of each agent.
										size_t *		pdAgentIndices,			// Out:	Indices of each agent.
										size_t *		pdCellIndices,			// Out:	Indices of the cell each agent is in.
										size_t const	numAgents//,				// In:	Number of agents in the simulation.
										//float3 const	worldSize				// In:	Extents of the world (for normalizing the positions).
										);

	__global__ void KNNBinningReorderData(	float3 const*	pdPosition,			// In: Agent positions.
											float3 const*	pdDirection,		// In: Agent directions.
											float const*	pdSpeed,			// In: Agent speeds.
					
											uint const*		pdAgentIndices,		// In: (sorted) agent index.
											uint const*		pdCellIndices,		// In: (sorted) cell index agent is in.

											float3 *		pdPositionSorted,	// Out: Sorted agent positions.
											float3 *		pdDirectionSorted,	// Out: Sorted agent directions.
											float *			pdSpeedSorted,		// Out: Sorted agent speeds.

											uint *			pdCellStart,		// Out: Start index of this cell in pdCellIndices.
											uint *			pdCellEnd,			// Out: End index of this cell in pdCellIndices.

											size_t const	numAgents
											);

	__global__ void KNNBinningKernel(	float3 const*	pdPositionSorted,	// In:	(sorted) Agent positions.

										uint const*		pdAgentIndices,		// In:	(sorted) Indices of each agent.
										uint const*		pdCellIndices,		// In:	(sorted) Indices of the cell each agent is in.
									
										uint const*		pdCellStart,		// In:	Start index of each cell in pdCellIndices.
										uint const*		pdCellEnd,			// In:	End index of each cell in pdCellIndices.

										uint *			pdKNNIndices,		// Out:	Indices of K Nearest Neighbors in pdPosition.
										float *			pdKNNDistances,		// Out:	Distances of the K Nearest Neighbors in pdPosition.

										size_t const	k,					// In:	Number of neighbors to consider.
										size_t const	radius,				// In:	Maximum radius (in cells) to consider.
										size_t const	numAgents			// In:	Number of agents in the simulation.
										);
}

__host__ void KNNBinningCUDABindTexture( hipArray * pdCudaArray )
{
	static hipChannelFormatDesc const channelDesc = hipCreateChannelDesc< uint >();

	texCellIndicesNormalized.normalized = true;
	texCellIndicesNormalized.filterMode = hipFilterModePoint;
	// Clamp out of bounds coordinates to the edge of the texture.
	texCellIndicesNormalized.addressMode[0] = hipAddressModeClamp;
	texCellIndicesNormalized.addressMode[1] = hipAddressModeClamp;
	texCellIndicesNormalized.addressMode[2] = hipAddressModeClamp;

	CUDA_SAFE_CALL( hipBindTextureToArray( texCellIndicesNormalized, pdCudaArray, channelDesc ) );

	texCellIndices.normalized = false;
	texCellIndices.filterMode = hipFilterModePoint;
	texCellIndices.addressMode[0] = hipAddressModeClamp;
	texCellIndices.addressMode[1] = hipAddressModeClamp;
	texCellIndices.addressMode[2] = hipAddressModeClamp;

	CUDA_SAFE_CALL( hipBindTextureToArray( texCellIndices, pdCudaArray, channelDesc ) );
}

__host__ void KNNBinningCUDAUnbindTexture( void )
{
	CUDA_SAFE_CALL( hipUnbindTexture( texCellIndicesNormalized ) );
	CUDA_SAFE_CALL( hipUnbindTexture( texCellIndices ) );
}

__inline__ __device__ bool WithinBounds( float3 const* point )
{
	return !(	point->x < constWorldMin.x || point->x > constWorldMax.x ||
				point->y < constWorldMin.y || point->y > constWorldMax.y ||
				point->z < constWorldMin.z || point->z > constWorldMax.z
				);
}

__device__ void GetNeighboringCells3D(	float3 const&	position,	// In:	Position to find neighbors for.
										uint *			pdCells,	// Out:	Array of cell indices.
										uint const&		r			// In:	Radius.
													)
{
	float3	queryPosition;
	uint	cellIndex = UINT_MAX;

	for( int dz = -r, iz = 0; dz <= r; dz++, iz++ )
	{
		for( int dy = -r, iy = 0; dy <= r; dy++, iy++ )
		{
			for( int dx = -r, ix = 0; dx <= r; dx++, ix++ )
			{
				queryPosition.x = position.x + dx * constWorldStep.x;
				queryPosition.y = position.y + dy * constWorldStep.y;
				queryPosition.z = position.z + dz * constWorldStep.z;

				if( WithinBounds( &queryPosition ) )
				{
					// Normalize the position.
					queryPosition.x = (queryPosition.x + 0.5f * constWorldSize.x) / constWorldSize.x;
					queryPosition.y = (queryPosition.y + 0.5f * constWorldSize.y) / constWorldSize.y;
					queryPosition.z = (queryPosition.z + 0.5f * constWorldSize.z) / constWorldSize.z;

					cellIndex = CELL_INDEX_NORMALIZED( queryPosition );
				}

				// Write the cell index to the output array.
				pdCells[iz*iy*r + iy*r + ix] = cellIndex;
			}
		}
	}
}

__device__ void GetNeighboringCells2D(	float3 const&	position,	// In:	Position to find neighbors for.
										uint *			pdCells,	// Out:	Array of cell indices.
										uint const&		r			// In:	Radius.
										)
{
	float3	queryPosition;
	uint	cellIndex = UINT_MAX;

	queryPosition.z = position.z;
	queryPosition.z = (queryPosition.z + 0.5f * constWorldSize.z) / constWorldSize.z;

	for( int dy = -r, iy = 0; dy <= r; dy++, iy++ )
	{
		for( int dx = -r, ix = 0; dx <= r; dx++, ix++ )
		{
			queryPosition.x = position.x + dx * constWorldStep.x;
			queryPosition.y = position.y + dy * constWorldStep.y;

			if( WithinBounds( &queryPosition ) )
			{
				// Normalize the position.
				queryPosition.x = (queryPosition.x + 0.5f * constWorldSize.x) / constWorldSize.x;
				queryPosition.y = (queryPosition.y + 0.5f * constWorldSize.y) / constWorldSize.y;

				cellIndex = CELL_INDEX_NORMALIZED( queryPosition );
			}

			// Don't add the same cell twice. Necessary check if the cell sizes are non-uniform.
			bool bContained = false;
			int const index = iy*r + ix;
			for( uint i = 0; i < index; i++ )
			{
				if( pdCells[i] == cellIndex )
					bContained = true;
			}
			if( bContained )
				continue;

			// Write the cell index to the output array.
			pdCells[ index ] = cellIndex;
		}
	}
}

__global__ void KNNBinningBuildDB(	float3 const*	pdPosition,				// In:	Positions of each agent.
									size_t *		pdAgentIndices,			// Out:	Indices of each agent.
									size_t *		pdCellIndices,			// Out:	Indices of the cell each agent is in.
									size_t const	numAgents//,				// In:	Number of agents in the simulation.
									//float3 const	worldSize				// In:	Extents of the world (for normalizing the positions).
									)
{
	// Offset of this agent in the global array.
	int const index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( index >= numAgents )
		return;

	// Copy the positions to shared memory.
	__shared__ float3 shPosition[THREADSPERBLOCK];
	FLOAT3_GLOBAL_READ( shPosition, pdPosition );

	__syncthreads();

	// Normalize the positions.
	//POSITION_SH( threadIdx.x ) = make_float3(	(POSITION_SH( threadIdx.x ).x + 0.5f * worldSize.x) / worldSize.x, 
	//											(POSITION_SH( threadIdx.x ).y + 0.5f * worldSize.y) / worldSize.y,
	//											(POSITION_SH( threadIdx.x ).z + 0.5f * worldSize.z) / worldSize.z );

	// Normalize the positions.
	POSITION_SH( threadIdx.x ).x = (POSITION_SH( threadIdx.x ).x + 0.5f * constWorldSize.x) / constWorldSize.x;
	//POSITION_SH( threadIdx.x ).y = (POSITION_SH( threadIdx.x ).y + 0.5f * constWorldSize.y) / constWorldSize.y;
	POSITION_SH( threadIdx.x ).z = (POSITION_SH( threadIdx.x ).z + 0.5f * constWorldSize.z) / constWorldSize.z;

	// Write the agent's cell index out to global memory.
	// TESTME: is this getting the right cell index?
	//pdCellIndices[index] = CELL_INDEX_NORMALIZED( POSITION_SH( threadIdx.x ) );

	pdCellIndices[index] = tex3D( texCellIndicesNormalized, POSITION_SH( threadIdx.x ).x, POSITION_SH( threadIdx.x ).y, POSITION_SH( threadIdx.x ).z );
	__syncthreads();

	// Write the agent's index out to global memory.
	pdAgentIndices[index] = index;
	__syncthreads();
}


__global__ void KNNBinningReorderData(	float3 const*	pdPosition,			// In: Agent positions.
										float3 const*	pdDirection,		// In: Agent directions.
										float const*	pdSpeed,			// In: Agent speeds.
				
										uint const*		pdAgentIndices,		// In: (sorted) agent index.
										uint const*		pdCellIndices,		// In: (sorted) cell index agent is in.

										float3 *		pdPositionSorted,	// Out: Sorted agent positions.
										float3 *		pdDirectionSorted,	// Out: Sorted agent directions.
										float *			pdSpeedSorted,		// Out: Sorted agent speeds.

										uint *			pdCellStart,		// Out: Start index of this cell in pdCellIndices.
										uint *			pdCellEnd,			// Out: End index of this cell in pdCellIndices.

										size_t const	numAgents
										)
{
	// Offset of this agent.
	int const index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( index >= numAgents )
		return;

	__shared__ uint shCellIndices[THREADSPERBLOCK+1];

	// Shared memory so we can coalesce the writes of sorted data to global memory.
	__shared__ float3 shPositionSorted[THREADSPERBLOCK];
	__shared__ float3 shDirectionSorted[THREADSPERBLOCK];
	__shared__ float shSpeedSorted[THREADSPERBLOCK];

	// Read the cell index of this agent.
	uint iCellIndex = pdCellIndices[ index ];
	__syncthreads();
	
	// Store cell index data in shared memory so that we can look 
	// at the neighboring agent's value without two reads per thread.
	shCellIndices[ threadIdx.x + 1 ] = iCellIndex;

	if( index > 0 && threadIdx.x == 0 )
	{
		// First thread in block must load neighbor agent cell index.
		shCellIndices[0] = pdCellIndices[ index - 1 ];
	}

	__syncthreads();

	// If this agent has a different cell index to the previous
	// agent then it must be the first in the cell,
	// so store the index of this agent in the cell.
	// As it isn't the first agent, it must also be the cell end of
	// the previous particle's cell

	if( index == 0 || iCellIndex != shCellIndices[ threadIdx.x ] )
	{
		pdCellStart[ iCellIndex ] = index;
		if( index > 0 )
			pdCellEnd[ shCellIndices[ threadIdx.x ] ] = index;
	}

	// If this is the last agent, the end index for the cell will be index + 1
	if( index == (numAgents - 1) )
	{
		pdCellEnd[ iCellIndex ] = index + 1;
	}

	// Use the sorted index to reorder the position/direction/speed data.
	uint const iSortedIndex = pdAgentIndices[ index ];

	// TODO:	This is potentially a lot faster using texture memory for the input data due to the 'random' nature of the access.
	//			Will require the transition to float4 instead of float3 to store the data though.
	shPositionSorted[ threadIdx.x ] = pdPosition[ iSortedIndex ];
	shDirectionSorted[ threadIdx.x ] = pdDirection[ iSortedIndex ];
	shSpeedSorted[ threadIdx.x ] = pdSpeed[ iSortedIndex ];

	__syncthreads();

	// Write to global memory.
	pdSpeedSorted[ index ] = shSpeedSorted[ threadIdx.x ];
	__syncthreads();
	
	FLOAT3_GLOBAL_WRITE( pdPositionSorted, shPositionSorted );
	FLOAT3_GLOBAL_WRITE( pdDirectionSorted, shDirectionSorted );
}

__global__ void KNNBinningKernel(	float3 const*	pdPositionSorted,	// In:	(sorted) Agent positions.

									uint const*		pdAgentIndices,		// In:	(sorted) Indices of each agent.
									uint const*		pdCellIndices,		// In:	(sorted) Indices of the cell each agent is currently in.
								
									uint const*		pdCellStart,		// In:	Start index of each cell in pdCellIndices.
									uint const*		pdCellEnd,			// In:	End index of each cell in pdCellIndices.

									uint *			pdKNNIndices,		// Out:	Indices of K Nearest Neighbors in pdPosition.
									float *			pdKNNDistances,		// Out:	Distances of the K Nearest Neighbors in pdPosition.

									size_t const	k,					// In:	Number of neighbors to consider.
									size_t const	radius,				// In:	Maximum radius (in cells) to consider.
									size_t const	numAgents			// In:	Number of agents in the simulation.
									)
{
	// Offset of this agent.
	int const index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( index >= numAgents )
		return;

	// Shared memory for local priority queue computations.
	extern __shared__ uint shKNNIndices[];
	float * shKNNDistances = (float*)shKNNIndices + THREADSPERBLOCK * k;

	// Set all elements of shKNNDistances to FLT_MAX, shKNNIndices to UINT_MAX.
	for( uint i = 0; i < k; i++ )
	{
		shKNNIndices[(threadIdx.x * k) + i] = UINT_MAX;
		shKNNDistances[(threadIdx.x * k) + i] = FLT_MAX;
	}

	// Store this thread's agent index and cell index in registers.
	uint const		agentIndex = pdAgentIndices[ index ];
	//uint const		cellIndex = pdCellIndices[ index ];
	float3 const	agentPosition = pdPositionSorted[ index ];

	//
	// TODO: for each surrounding cell within radius...
	//

	// TODO: support for varying radius.
	__shared__ uint shNeighboringCells[9*THREADSPERBLOCK];
	GetNeighboringCells2D( agentPosition, &shNeighboringCells[threadIdx.x*9], 1 );

	uint cellIndex = UINT_MAX;
	for( uint i = 0; i < 9; i++ )
	{
		cellIndex = shNeighboringCells[threadIdx.x*9+i];
		if( UINT_MAX == cellIndex )
			continue;

		// For each agent in the cell...
		for( uint otherIndexSorted = pdCellStart[ cellIndex ]; otherIndexSorted < pdCellEnd[ cellIndex ]; otherIndexSorted++ )
		{
			// Get the index of the other agent (unsorted).
			uint const otherIndex = pdAgentIndices[ otherIndexSorted ];
			
			// Do not include self.
			if( agentIndex == otherIndex )
				continue;

			// Compute the distance between this agent and the one at i.
			// TODO: texture memory....
			float const dist = float3_distance( agentPosition, pdPositionSorted[ otherIndexSorted ] );

			if( dist < shKNNDistances[(threadIdx.x * k) + (k - 1)] )	// Distance of the kth closest agent.
			{
				// Agent at index i is the new (at least) kth closest. Set the distance and index in shared mem.
				shKNNDistances[(threadIdx.x * k) + (k - 1)] = dist;
				shKNNIndices[(threadIdx.x * k) + (k - 1)] = otherIndex;

				// Bubble the values up...
				for( int slot = k - 2; slot >= 0; slot-- )
				{
					if( shKNNDistances[(threadIdx.x * k) + slot] > shKNNDistances[(threadIdx.x * k) + (slot + 1)] )
					{
						swap( shKNNDistances[(threadIdx.x * k) + slot], shKNNDistances[(threadIdx.x * k) + (slot + 1)] );
						swap( shKNNIndices[(threadIdx.x * k) + slot], shKNNIndices[(threadIdx.x * k) + (slot + 1)] );
					}
					else
						break;
				}
			}
		}
	}

	__syncthreads();

	// Write the shKNNIndices and shKNNDistances values out to global memory.
	for( uint i = 0; i < k; i++ )
	{
		pdKNNIndices[agentIndex*k + i] = shKNNIndices[threadIdx.x*k + i];
		pdKNNDistances[agentIndex*k + i] = shKNNDistances[threadIdx.x*k + i];
	}
	__syncthreads();
}
