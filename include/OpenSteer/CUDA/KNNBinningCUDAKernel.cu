#include "hip/hip_runtime.h"
#include "KNNBinningCUDA.cuh"

#include "../VectorUtils.cuh"

#include "CUDAKernelGlobals.cuh"

using namespace OpenSteer;

// Define the texture reference to access the appropriate bin_cell's index.
texture< uint, hipTextureType3D, hipReadModeElementType > texCellIndicesNormalized;
texture< uint, hipTextureType3D, hipReadModeElementType > texCellIndices;

// Constant memory used to hold the worldSize and worldCells values.
__constant__ float3		constWorldSize;
__constant__ float3		constWorldStep;
__constant__ float3		constWorldStepNormalized;
__constant__ uint3		constWorldCells;
//__constant__ float3		constWorldMin;
//__constant__ float3		constWorldMax;

// Fetch the cell index from texCellIndicesNormalized at a given world {x,y,z} position.
#define CELL_INDEX_NORMALIZED( pos )	( tex3D( texCellIndicesNormalized, pos.x, pos.y, pos.z ) )
// Fetch the cell index from texCellIndices at a given texel (x,y,z) coordinate.
#define CELL_INDEX( x, y, z )			( tex3D( texCellIndices, x, y, z ) )

// Kernel declarations.
extern "C"
{
	// Bind the textures to the input hipArray.
	__host__ void KNNBinningCUDABindTexture( hipArray * pCudaArray );
	// Unbind the textures.
	__host__ void KNNBinningCUDAUnbindTexture( void );

	// Kernel to set initial bin indices of vehicles in the simulation.
	__global__ void KNNBinningBuildDB(	float3 const*	pdPosition,				// In:	Positions of each agent.
										float3 *		pdPositionNormalized,	// Out:	Normalized positions of each agent.
										size_t *		pdAgentIndices,			// Out:	Indices of each agent.
										size_t *		pdCellIndices,			// Out:	Indices of the cell each agent is in.
										size_t const	numAgents				// In:	Number of agents in the simulation.
										);

	__global__ void KNNBinningReorderData(	float3 const*	pdPosition,					// In: Agent positions.
											float3 const*	pdPositionNormalized,		// In:	Normalized agent positions.
											float3 const*	pdDirection,				// In: Agent directions.
											float const*	pdSpeed,					// In: Agent speeds.
					
											uint const*		pdAgentIndices,				// In: (sorted) agent index.
											uint const*		pdCellIndices,				// In: (sorted) cell index agent is in.

											float3 *		pdPositionSorted,			// Out: Sorted agent positions.
											float3 *		pdPositionNormalizedSorted,	// Out:	Sorted normalized agent positions.
											float3 *		pdDirectionSorted,			// Out: Sorted agent directions.
											float *			pdSpeedSorted,				// Out: Sorted agent speeds.

											uint *			pdCellStart,				// Out: Start index of this cell in pdCellIndices.
											uint *			pdCellEnd,					// Out: End index of this cell in pdCellIndices.

											size_t const	numAgents
											);

	__global__ void KNNBinningKernel(	float3 const*	pdPositionSorted,			// In:	Sorted agent positions.
										float3 const*	pdPositionNormalizedSorted,	// In:	Sorted normalized agent positions.

										uint const*		pdAgentIndices,				// In:	Sorted agent indices.
										uint const*		pdCellIndices,				// In:	Sorted indices of the cell each agent is currently in.
									
										uint const*		pdCellStart,				// In:	Start index of each cell in pdCellIndices.
										uint const*		pdCellEnd,					// In:	End index of each cell in pdCellIndices.

										uint *			pdKNNIndices,				// Out:	Indices of K Nearest Neighbors in pdPosition.
										float *			pdKNNDistances,				// Out:	Distances of the K Nearest Neighbors in pdPosition.

										size_t const	k,							// In:	Number of neighbors to consider.
										size_t const	radius,						// In:	Maximum radius (in cells) to consider.
										size_t const	numAgents					// In:	Number of agents in the simulation.
										);
}

__host__ void KNNBinningCUDABindTexture( hipArray * pdCudaArray )
{
	static hipChannelFormatDesc const channelDesc = hipCreateChannelDesc< uint >();

	texCellIndicesNormalized.normalized = true;
	texCellIndicesNormalized.filterMode = hipFilterModePoint;
	// Clamp out of bounds coordinates to the edge of the texture.
	texCellIndicesNormalized.addressMode[0] = hipAddressModeClamp;
	texCellIndicesNormalized.addressMode[1] = hipAddressModeClamp;
	texCellIndicesNormalized.addressMode[2] = hipAddressModeClamp;

	CUDA_SAFE_CALL( hipBindTextureToArray( texCellIndicesNormalized, pdCudaArray, channelDesc ) );

	texCellIndices.normalized = false;
	texCellIndices.filterMode = hipFilterModePoint;
	texCellIndices.addressMode[0] = hipAddressModeClamp;
	texCellIndices.addressMode[1] = hipAddressModeClamp;
	texCellIndices.addressMode[2] = hipAddressModeClamp;

	CUDA_SAFE_CALL( hipBindTextureToArray( texCellIndices, pdCudaArray, channelDesc ) );
}

__host__ void KNNBinningCUDAUnbindTexture( void )
{
	CUDA_SAFE_CALL( hipUnbindTexture( texCellIndicesNormalized ) );
	CUDA_SAFE_CALL( hipUnbindTexture( texCellIndices ) );
}

//__inline__ __device__ bool WithinBounds( float3 const* point )
//{
//	return !(	point->x < constWorldMin.x || point->x > constWorldMax.x ||
//				point->y < constWorldMin.y || point->y > constWorldMax.y ||
//				point->z < constWorldMin.z || point->z > constWorldMax.z
//				);
//}

__device__ void GetCellsWithinRadius3D(	float3 const&	positionNormalized,	// In:	Normalized position to find neighboring cells for.
										uint *			pdCells,			// Out:	Array of cell indices.
										int const&		r					// In:	Radius.
													)
{
	for( int dz = -r, iz = 0; dz <= r; dz++, iz++ )
	{
		for( int dy = -r, iy = 0; dy <= r; dy++, iy++ )
		{
			for( int dx = -r, ix = 0; dx <= r; dx++, ix++ )
			{
				int const index = iz*iy*(r*2+1) + iy*(r*2+1) + ix;
				pdCells[ index ] = UINT_MAX;

				// Compute the query position for this iteration.
				float3	queryPosition = make_float3(	positionNormalized.x + dx * constWorldStepNormalized.x,
														positionNormalized.y + dy * constWorldStepNormalized.y,
														positionNormalized.z + dz * constWorldStepNormalized.z
														);

				// Get the cell index for the query position.
				uint const cellIndex = CELL_INDEX_NORMALIZED( queryPosition );

				// Don't add the same cell twice. Will be a necessary check if the cell sizes are non-uniform.
				bool bContained = false;
				for( uint i = 0; i < index; i++ )
				{
					if( pdCells[i] == cellIndex )
						bContained = true;
				}
				if( bContained )
					continue;

				// Write the cell index to the output array.
				pdCells[ index ] = cellIndex;
			}
		}
	}
}

__device__ void GetCellsWithinRadius2D(	float3 const&	positionNormalized,	// In:	Position to find neighbors for.
										uint *			pdCells,			// Out:	Array of cell indices.
										int const&		r					// In:	Radius.
										)
{
	for( int dz = -r, iz = 0; dz <= r; dz++, iz++ )
	{
		for( int dx = -r, ix = 0; dx <= r; dx++, ix++ )
		{
			int const index = iz*(r*2+1) + ix;
			// Set the cell index to UINT_MAX.
			pdCells[ index ] = UINT_MAX;

			// Compute the query position for this iteration.
			float3	queryPosition = make_float3(	positionNormalized.x + dx * constWorldStepNormalized.x,
													positionNormalized.y,
													positionNormalized.z + dz * constWorldStepNormalized.z
													);

			// Get the cell index for the query position.
			uint const cellIndex = CELL_INDEX_NORMALIZED( queryPosition );

			// Don't add the same cell twice. Will be a necessary check if the cell sizes are non-uniform.
			bool bContained = false;
			for( uint i = 0; i < index; i++ )
			{
				if( pdCells[i] == cellIndex )
					bContained = true;
			}
			if( bContained )
				continue;

			// Write the cell index to the output array.
			pdCells[ index ] = cellIndex;
		}
	}
}

__global__ void KNNBinningBuildDB(	float3 const*	pdPosition,				// In:	Positions of each agent.
									float3 *		pdPositionNormalized,	// Out:	Normalized positions of each agent.
									size_t *		pdAgentIndices,			// Out:	Indices of each agent.
									size_t *		pdCellIndices,			// Out:	Indices of the cell each agent is in.
									size_t const	numAgents				// In:	Number of agents in the simulation.
									)
{
	// Offset of this agent in the global array.
	int const index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( index >= numAgents )
		return;

	// Copy the positions to shared memory.
	__shared__ float3 shPosition[THREADSPERBLOCK];
	FLOAT3_GLOBAL_READ( shPosition, pdPosition );

	__syncthreads();

	// Normalize the positions.
	POSITION_SH( threadIdx.x ).x = (POSITION_SH( threadIdx.x ).x + 0.5f * constWorldSize.x) / constWorldSize.x;
	//POSITION_SH( threadIdx.x ).y = (POSITION_SH( threadIdx.x ).y + 0.5f * constWorldSize.y) / constWorldSize.y;	// TODO: effect of normalizing y?
	POSITION_SH( threadIdx.x ).z = (POSITION_SH( threadIdx.x ).z + 0.5f * constWorldSize.z) / constWorldSize.z;

	// Write the normalized positions to global memory.
	FLOAT3_GLOBAL_WRITE( pdPositionNormalized, shPosition );

	pdCellIndices[index] = tex3D( texCellIndicesNormalized, POSITION_SH( threadIdx.x ).x, POSITION_SH( threadIdx.x ).y, POSITION_SH( threadIdx.x ).z );
	__syncthreads();

	// Write the agent's index out to global memory.
	pdAgentIndices[index] = index;
	__syncthreads();
}


__global__ void KNNBinningReorderData(	float3 const*	pdPosition,					// In:	Agent positions.
										float3 const*	pdPositionNormalized,		// In:	Normalized agent positions.
										float3 const*	pdDirection,				// In:	Agent directions.
										float const*	pdSpeed,					// In:	Agent speeds.
				
										uint const*		pdAgentIndices,				// In:	(sorted) agent index.
										uint const*		pdCellIndices,				// In:	(sorted) cell index agent is in.

										float3 *		pdPositionSorted,			// Out:	Sorted agent positions.
										float3 *		pdPositionNormalizedSorted,	// Out:	Sorted agent positions.
										float3 *		pdDirectionSorted,			// Out:	Sorted agent directions.
										float *			pdSpeedSorted,				// Out:	Sorted agent speeds.

										uint *			pdCellStart,				// Out:	Start index of this cell in pdCellIndices.
										uint *			pdCellEnd,					// Out:	End index of this cell in pdCellIndices.

										size_t const	numAgents
										)
{
	// Offset of this agent.
	int const index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( index >= numAgents )
		return;

	__shared__ uint shCellIndices[THREADSPERBLOCK+1];

	// Shared memory so we can coalesce the writes of sorted data to global memory.
	__shared__ float3 shPositionSorted[THREADSPERBLOCK];
	__shared__ float3 shPositionNormalizedSorted[THREADSPERBLOCK];
	__shared__ float3 shDirectionSorted[THREADSPERBLOCK];
	__shared__ float shSpeedSorted[THREADSPERBLOCK];

	// Read the cell index of this agent.
	uint iCellIndex = pdCellIndices[ index ];
	__syncthreads();
	
	// Store cell index data in shared memory so that we can look 
	// at the neighboring agent's value without two reads per thread.
	shCellIndices[ threadIdx.x + 1 ] = iCellIndex;

	if( index > 0 && threadIdx.x == 0 )
	{
		// First thread in block must load neighbor agent cell index.
		shCellIndices[0] = pdCellIndices[ index - 1 ];
	}

	__syncthreads();

	// If this agent has a different cell index to the previous
	// agent then it must be the first in the cell,
	// so store the index of this agent in the cell.
	// As it isn't the first agent, it must also be the cell end of
	// the previous particle's cell

	if( index == 0 || iCellIndex != shCellIndices[ threadIdx.x ] )
	{
		pdCellStart[ iCellIndex ] = index;
		if( index > 0 )
			pdCellEnd[ shCellIndices[ threadIdx.x ] ] = index;
	}

	// If this is the last agent, the end index for the cell will be index + 1
	if( index == (numAgents - 1) )
	{
		pdCellEnd[ iCellIndex ] = index + 1;
	}

	// Use the sorted index to reorder the position/direction/speed data.
	uint const iSortedIndex = pdAgentIndices[ index ];

	// TODO:	This is potentially a lot faster using texture memory for the input data due to the 'random' nature of the access.
	//			Will require the transition to float4 instead of float3 to store the data though.
	shPositionSorted[ threadIdx.x ] = pdPosition[ iSortedIndex ];
	shDirectionSorted[ threadIdx.x ] = pdDirection[ iSortedIndex ];
	shSpeedSorted[ threadIdx.x ] = pdSpeed[ iSortedIndex ];
	shPositionNormalizedSorted[ threadIdx.x ] = pdPositionNormalized[ iSortedIndex ];

	__syncthreads();

	// Write to global memory.
	pdSpeedSorted[ index ] = shSpeedSorted[ threadIdx.x ];
	__syncthreads();
	
	FLOAT3_GLOBAL_WRITE( pdPositionSorted, shPositionSorted );
	FLOAT3_GLOBAL_WRITE( pdDirectionSorted, shDirectionSorted );
	FLOAT3_GLOBAL_WRITE( pdPositionNormalizedSorted, shPositionNormalizedSorted );
}

__global__ void KNNBinningKernel(	float3 const*	pdPositionSorted,			// In:	(sorted) Agent positions.
									float3 const*	pdPositionNormalizedSorted,

									uint const*		pdAgentIndices,				// In:	(sorted) Indices of each agent.
									uint const*		pdCellIndices,				// In:	(sorted) Indices of the cell each agent is currently in.
								
									uint const*		pdCellStart,				// In:	Start index of each cell in pdCellIndices.
									uint const*		pdCellEnd,					// In:	End index of each cell in pdCellIndices.

									uint *			pdKNNIndices,				// Out:	Indices of K Nearest Neighbors in pdPosition.
									float *			pdKNNDistances,				// Out:	Distances of the K Nearest Neighbors in pdPosition.

									size_t const	k,							// In:	Number of neighbors to consider.
									size_t const	radius,						// In:	Maximum radius (in cells) to consider.
									size_t const	numAgents					// In:	Number of agents in the simulation.
									)
{
	// Offset of this agent.
	int const index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( index >= numAgents )
		return;

	// Shared memory for local priority queue computations.
	extern __shared__ uint shKNNIndices[];
	float * shKNNDistances = (float*)shKNNIndices + THREADSPERBLOCK * k;

	// Set all elements of shKNNDistances to FLT_MAX, shKNNIndices to UINT_MAX.
	for( uint i = 0; i < k; i++ )	// TODO: test effect of unrolling.
	{
		shKNNIndices[(threadIdx.x * k) + i] = UINT_MAX;
		shKNNDistances[(threadIdx.x * k) + i] = FLT_MAX;
	}

	// Store this thread's agent index and cell index in registers.
	uint const		agentIndex = pdAgentIndices[ index ];
	//uint const		cellIndex = pdCellIndices[ index ];
	float3 const	agentPosition = pdPositionSorted[ index ];
	float3 const	agentPositionNormalized = pdPositionNormalizedSorted[ index ];

	//
	// TODO: for each surrounding cell within radius...
	//

	// TODO: support for varying radius.
	__shared__ uint shNeighboringCells[9*THREADSPERBLOCK];
	GetCellsWithinRadius2D( agentPositionNormalized, &shNeighboringCells[threadIdx.x*9], radius );

	uint cellIndex = UINT_MAX;
	for( uint i = 0; i < 9; i++ )
	{
		cellIndex = shNeighboringCells[threadIdx.x*9+i];
		if( UINT_MAX == cellIndex )
			continue;

		// For each agent in the cell...
		for( uint otherIndexSorted = pdCellStart[ cellIndex ]; otherIndexSorted < pdCellEnd[ cellIndex ]; otherIndexSorted++ )
		{
			// Get the index of the other agent (unsorted).
			uint const otherIndex = pdAgentIndices[ otherIndexSorted ];
			
			// Do not include self.
			if( agentIndex == otherIndex )
				continue;

			// Compute the distance between this agent and the one at i.
			// TODO: texture memory....
			float const dist = float3_distance( agentPosition, pdPositionSorted[ otherIndexSorted ] );

			if( dist < shKNNDistances[(threadIdx.x * k) + (k - 1)] )	// Distance of the kth closest agent.
			{
				// Agent at index i is the new (at least) kth closest. Set the distance and index in shared mem.
				shKNNDistances[(threadIdx.x * k) + (k - 1)] = dist;
				shKNNIndices[(threadIdx.x * k) + (k - 1)] = otherIndex;

				// Bubble the values up...
				for( int slot = k - 2; slot >= 0; slot-- )
				{
					if( shKNNDistances[(threadIdx.x * k) + slot] > shKNNDistances[(threadIdx.x * k) + (slot + 1)] )
					{
						swap( shKNNDistances[(threadIdx.x * k) + slot], shKNNDistances[(threadIdx.x * k) + (slot + 1)] );
						swap( shKNNIndices[(threadIdx.x * k) + slot], shKNNIndices[(threadIdx.x * k) + (slot + 1)] );
					}
					else
						break;
				}
			}
		}
	}

	__syncthreads();

	// Write the shKNNIndices and shKNNDistances values out to global memory.
	for( uint i = 0; i < k; i++ )
	{
		pdKNNIndices[agentIndex*k + i] = shKNNIndices[threadIdx.x*k + i];
		pdKNNDistances[agentIndex*k + i] = shKNNDistances[threadIdx.x*k + i];
	}
	__syncthreads();
}
