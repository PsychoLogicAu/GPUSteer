#include "hip/hip_runtime.h"
#include "SteerToAvoidNeighborsCUDA.cuh"

using namespace OpenSteer;

extern "C"
{
	__global__ void SteerToAvoidNeighborsCUDAKernel(	uint const*		pdKNNIndices,			// In:		Indices of the KNN for each agent.
														float const*	pdKNNDistances,			// In:		Distances to the KNN for each agent.
														size_t const	k,						// In:		Number of KNN for each agent.

														// Group A data.
														float3 const*	pdAPosition,				// In:		Positions of each agent.
														float3 const*	pdADirection,			// In:		Directions of facing for each agent.
														float const*	pdARadius,				// In:		Radius of each agent.
														float3 const*	pdASide,					// In:		Side direction for each agent.

														float *			pdASpeed,				// In/Out:	Speed of each agent.
														float3 *		pdASteering,				// Out:		Steering vectors for each agent.

														// Group B data.
														float3 const*	pdBPosition,
														float3 const*	pdBDirection,
														float const*	pdBSpeed,
														float const*	pdBRadius,


														float const		minTimeToCollision,		// In:		Look-ahead time for collision avoidance.
														float const		minSeparationDistance,	// In:		Distance to consider 'close' neighbors.

														size_t const	numAgents,
														float const		fWeight,

														uint *			pdAppliedKernels,
														uint const		doNotApplyWith
														);
}

SteerToAvoidNeighborsCUDA::SteerToAvoidNeighborsCUDA( AgentGroup * pAgentGroup, KNNData * pKNNData, AgentGroup * pOtherGroup, float const fMinTimeToCollision, float const fMinSeparationDistance, float const fWeight, uint const doNotApplyWith )
:	AbstractCUDAKernel( pAgentGroup, fWeight, doNotApplyWith ),
	m_pKNNData( pKNNData ),
	m_pOtherGroup( pOtherGroup ),
	m_fMinTimeToCollision( fMinTimeToCollision ),
	m_fMinSeparationDistance( fMinSeparationDistance )
{
}

void SteerToAvoidNeighborsCUDA::init( void )
{
	// Nothing to do.
}

void SteerToAvoidNeighborsCUDA::run( void )
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	size_t const&	k					= m_pKNNData->k();
	size_t const&	numAgents			= getNumAgents();

	// Gather the required device pointers.
	uint const*		pdKNNIndices		= m_pKNNData->pdKNNIndices();
	float const*	pdKNNDistances		= m_pKNNData->pdKNNDistances();

	float3 const*	pdAPosition			= m_pAgentGroupData->pdPosition();
	float3 const*	pdADirection		= m_pAgentGroupData->pdForward();
	float3 const*	pdASide				= m_pAgentGroupData->pdSide();
	float *			pdASpeed			= m_pAgentGroupData->pdSpeed();

	float const*	pdARadius			= m_pAgentGroupConst->pdRadius();

	float3 *		pdASteering			= m_pAgentGroupData->pdSteering();

	float3 const*	pdBPosition			= m_pOtherGroup->pdPosition();
	float3 const*	pdBDirection		= m_pOtherGroup->pdDirection();
	float const*	pdBSpeed			= m_pOtherGroup->pdSpeed();
	float const*	pdBRadius			= m_pOtherGroup->pdRadius();

	uint *			pdAppliedKernels	= m_pAgentGroupData->pdAppliedKernels();

	size_t shMemSize = k * THREADSPERBLOCK * (sizeof(uint) + sizeof(float));
	
	SteerToAvoidNeighborsCUDAKernel<<< grid, block, shMemSize >>>(	pdKNNIndices, pdKNNDistances, k,
																	pdAPosition, pdADirection, pdARadius, pdASide,
																	pdASpeed, pdASteering,
																	pdBPosition, pdBDirection, pdBSpeed, pdBRadius,
																	m_fMinTimeToCollision, m_fMinSeparationDistance,
																	numAgents, m_fWeight, pdAppliedKernels, m_doNotApplyWith
																	);
	cutilCheckMsg( "SteerToAvoidNeighborsCUDAKernel failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

void SteerToAvoidNeighborsCUDA::close(void )
{
	// Device data has changed. Instruct the AgentGroup it needs to synchronize the host.
	m_pAgentGroup->SetSyncHost();
}
