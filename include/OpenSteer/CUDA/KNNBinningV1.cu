#include "hip/hip_runtime.h"
#include "KNNBinningV1.cuh"

using namespace OpenSteer;

#include "KNNBinData.cuh"

#include <thrust/sort.h>

//#include "DebugUtils.h"

#define TIMING

// Kernel file function prototypes.
extern "C"
{
	// Bind texCellIndices to the hipArray.
	__host__ void KNNBinningV1BindTexture( hipArray * pCudaArray );
	__host__ void KNNBinningV1UnbindTexture( void );

	__host__ void KNNBinningV1KernelBindTextures(			uint const*		pdBCellStart,
														uint const*		pdBCellEnd,
														uint const*		pdBIndices,
														float4 const*	pdBPositionSorted,
														uint const		numCells,
														uint const		numB
														);

	__host__ void KNNBinningV1KernelUnbindTextures( void );
	__host__ void KNNBinningV1ReorderDBBindTextures(	float4 const*	pdPosition,
													uint const		numAgents );
	__host__ void KNNBinningV1ReorderDBUnbindTextures( void );

	__global__ void KNNBinningV1BuildDB(					float4 const*	pdPosition,				// In:	Positions of each agent.
														size_t *		pdAgentIndices,			// Out:	Indices of each agent.
														size_t *		pdCellIndices,			// Out:	Indices of the cell each agent is in.
														size_t const	numAgents
														);

	__global__ void KNNBinningV1ReorderDB(				uint const*		pdAgentIndices,		// In: (sorted) agent index.
														uint const*		pdCellIndices,		// In: (sorted) cell index agent is in.

														float4 *		pdPositionSorted,	// Out: Sorted agent positions.

														uint *			pdCellStart,		// Out: Start index of this cell in pdCellIndices.
														uint *			pdCellEnd,			// Out: End index of this cell in pdCellIndices.

														size_t const	numAgents
														);

	__global__ void KNNBinningV1Kernel(					// Group A
														float4 const*	pdAPositionSorted,			// In:	Sorted group A positions.
														uint const*		pdAIndices,					// In:	Sorted group A indices
														uint const		numA,						// In:	Size of group A.

														// Cell neighbor info.
														int const		radius,						// In:	Search radius (in cells) to consider.

														// Output data.
														uint *			pdKNNIndices,				// Out:	Indices of K Nearest Neighbors in pdPosition.
														float *			pdKNNDistances,				// Out:	Distances of the K Nearest Neighbors in pdPosition.
														uint const		k,							// In:	Number of neighbors to consider.

														uint const		numB,						// In:	Size of group B.
														bool const		groupWithSelf				// In:	Are we testing this group with itself? (group A == group B)
														);
}

#pragma region KNNBinningV1UpdateDBCUDA

KNNBinningV1UpdateDBCUDA::KNNBinningV1UpdateDBCUDA( BaseGroup * pGroup, KNNBinData * pKNNBinData )
:	AbstractCUDAKernel( NULL, 1.f, 0 ),
	m_pGroup( pGroup ),
	m_pKNNBinData( pKNNBinData )
{
	// Nothing to do.
}

void KNNBinningV1UpdateDBCUDA::init( void )
{
	// Bind the lookup texture.
	KNNBinningV1BindTexture( m_pKNNBinData->pdCellIndexArray() );
}

void KNNBinningV1UpdateDBCUDA::run( void )
{
	dim3 grid	= dim3( (m_pGroup->Size() + THREADSPERBLOCK - 1) / THREADSPERBLOCK );
	dim3 block	= dim3( THREADSPERBLOCK );

	// Gather required data.
	float4 const*	pdPosition				= m_pGroup->pdPosition();
	
	uint *			pdCellIndices			= m_pGroup->GetKNNDatabase().pdCellIndices();

	uint *			pdAgentIndicesSorted	= m_pGroup->GetKNNDatabase().pdAgentIndicesSorted();
	uint *			pdCellIndicesSorted		= m_pGroup->GetKNNDatabase().pdCellIndicesSorted();

	float4 *		pdPositionSorted		= m_pGroup->GetKNNDatabase().pdPositionSorted();
	uint *			pdCellStart				= m_pGroup->GetKNNDatabase().pdCellStart();
	uint *			pdCellEnd				= m_pGroup->GetKNNDatabase().pdCellEnd();

	uint const&		numAgents				= m_pGroup->Size();

#if defined TIMING
	//
	//	TIMING: hard to get exact times with profiling, too many operations.
	//
	// Events for timing the complete operation.
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );
#endif

	// Call KNNBinningBuildDB to build the database. 
	KNNBinningV1BuildDB<<< grid, block >>>( pdPosition, pdAgentIndicesSorted, pdCellIndices, numAgents );
	cutilCheckMsg( "KNNBinningBuildDB failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	// Copy pdCellIndices to  pdCellIndicesSorted.
	CUDA_SAFE_CALL( hipMemcpy( pdCellIndicesSorted, pdCellIndices, numAgents * sizeof(uint), hipMemcpyDeviceToDevice ) );

	// Sort pdAgentIndicesSorted on pdCellIndicesSorted using thrust.
	thrust::sort_by_key(	thrust::device_ptr<uint>( pdCellIndicesSorted ),
							thrust::device_ptr<uint>( pdCellIndicesSorted + numAgents ),
							thrust::device_ptr<uint>( pdAgentIndicesSorted ) );

	// Set all cells to empty.
	CUDA_SAFE_CALL( hipMemset( pdCellStart, 0xffffffff, m_pKNNBinData->getNumCells() * sizeof(uint) ) );

	// Bind the textures.
	KNNBinningV1ReorderDBBindTextures( pdPosition, numAgents );

	// Call KNNBinningReorderDB to re-order the data in the DB.
	KNNBinningV1ReorderDB<<< grid, block >>>( pdAgentIndicesSorted, pdCellIndicesSorted, pdPositionSorted, pdCellStart, pdCellEnd, numAgents );	cutilCheckMsg( "KNNBinningReorderDB failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	// Unbind the textures.
	KNNBinningV1ReorderDBUnbindTextures();

#if defined TIMING
	//
	//	TIMING:
	//
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	
	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );
	char szString[128] = {0};
	sprintf_s( szString, "KNNBinningV1UpdateDBCUDA,%f\n", elapsedTime );
	//OutputDebugStringToFile( szString );
	OutputDebugString( szString );

	// Destroy the events.
	hipEventDestroy( start );
	hipEventDestroy( stop );
#endif
}

void KNNBinningV1UpdateDBCUDA::close( void )
{
	// Unbind the texture.
	KNNBinningV1UnbindTexture();

	// The AgentGroup's database has now changed.
	m_pGroup->SetSyncHost();
}
#pragma endregion


#pragma region KNNBinningV1CUDA

KNNBinningV1CUDA::KNNBinningV1CUDA( AgentGroup * pAgentGroup, KNNData * pKNNData, KNNBinData * pKNNBinData, BaseGroup * pOtherGroup, uint const searchRadius )
:	AbstractCUDAKernel( pAgentGroup, 1.f, 0 ),
	m_pKNNData( pKNNData ),
	m_pKNNBinData( pKNNBinData ),
	m_pOtherGroup( pOtherGroup ),
	m_searchRadius( searchRadius )
{
}

void KNNBinningV1CUDA::init( void )
{
	// Bind the cell indices texture.
	//KNNBinningV1CUDABindTexture( m_pKNNBinData->pdCellIndexArray() );
}

void KNNBinningV1CUDA::run( void )
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	// Gather the required data.
	float4 const*		pdAPositionSorted		= m_pAgentGroup->GetKNNDatabase().pdPositionSorted();
	uint const*			pdAIndices				= m_pAgentGroup->GetKNNDatabase().pdAgentIndicesSorted();
	uint const&			numA					= getNumAgents();

	float4 const*		pdBPositionSorted		= m_pOtherGroup->GetKNNDatabase().pdPositionSorted();
	uint const*			pdBIndices				= m_pOtherGroup->GetKNNDatabase().pdAgentIndicesSorted();
	uint const&			numB					= m_pOtherGroup->Size();

	uint const*			pdBCellStart			= m_pOtherGroup->GetKNNDatabase().pdCellStart();
	uint const*			pdBCellEnd				= m_pOtherGroup->GetKNNDatabase().pdCellEnd();
	uint const&			numCells				= m_pKNNBinData->getNumCells();

	uint *				pdKNNIndices			= m_pKNNData->pdKNNIndices();
	float *				pdKNNDistances			= m_pKNNData->pdKNNDistances();
	uint const&			k						= m_pOtherGroup->GetKNNDatabase().k();

	bool const			groupWithSelf			= m_pAgentGroup == m_pOtherGroup;

	// Compute the size of shared memory needed for each block.
	size_t shMemSize = k * THREADSPERBLOCK * (sizeof(float) + sizeof(uint));

#if defined TIMING
	//
	//	TIMING: hard to get exact times with profiling, too many operations.
	//
	// Events for timing the complete operation.
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );
#endif

	// Bind the textures.
	KNNBinningV1KernelBindTextures(	pdBCellStart, pdBCellEnd, pdBIndices, pdBPositionSorted, numCells, numB );

	KNNBinningV1BindTexture( m_pKNNBinData->pdCellIndexArray() );

	// Call the KNNBinning kernel.
	KNNBinningV1Kernel<<< grid, block, shMemSize >>>(	pdAPositionSorted,
													pdAIndices,
													numA,
													m_searchRadius,
													pdKNNIndices,
													pdKNNDistances,
													k,
													numB,
													groupWithSelf
													);
	cutilCheckMsg( "KNNBinningKernel failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	// Unbind the textures.
	KNNBinningV1KernelUnbindTextures();
	KNNBinningV1UnbindTexture();

#if defined TIMING && defined _DEBUG
	//
	//	TIMING:
	//
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	
	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );
	char szString[128] = {0};
	sprintf_s( szString, "KNNBinningV1CUDA,%f\n", elapsedTime );
	//OutputDebugStringToFile( szString );
	OutputDebugString( szString );

	// Destroy the events.
	hipEventDestroy( start );
	hipEventDestroy( stop );
#endif
}

void KNNBinningV1CUDA::close( void )
{
	// The KNNData has most likely changed.
	m_pKNNData->setSyncHost();
}

#pragma endregion
