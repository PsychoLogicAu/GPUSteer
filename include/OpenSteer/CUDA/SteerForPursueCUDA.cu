#include "hip/hip_runtime.h"
#include "SteerForPursueCUDA.cuh"

#include "../AgentGroupData.cuh"

using namespace OpenSteer;

// Kernel function prototype.
extern "C"
{
__global__ void SteerForPursueCUDAKernel(	float4 const* pdPosition,
											float4 const* pdDirection,
											float const* pdSpeed, 

											float3 const targetPosition,
											float3 const targetForward,
											float3 const targetVelocity,
											float const targetSpeed,

											float4 * pdSteering,

											size_t const numAgents,
											float const maxPredictionTime,
											float const fWeight,
											uint * pdAppliedKernels,
											uint const doNotApplyWith
											);
}

SteerForPursueCUDA::SteerForPursueCUDA(	AgentGroup * pAgentGroup, 
										float3 const& targetPosition,
										float3 const& targetDirection,
										float const& targetSpeed,

										const float fMaxPredictionTime,
										float const fWeight,
										uint const doNotApplyWith
							)
:	AbstractCUDAKernel( pAgentGroup, fWeight, doNotApplyWith ),
	m_targetPosition( targetPosition ),
	m_targetDirection( targetDirection ),
	m_targetSpeed( targetSpeed ),
	m_fMaxPredictionTime( fMaxPredictionTime )
{
	m_targetVelocity = float3_scalar_multiply( m_targetDirection, m_targetSpeed );
}

void SteerForPursueCUDA::init(void)
{ }

void SteerForPursueCUDA::run(void)
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	// Gether the required device pointers.
	float4 *		pdSteering			= m_pAgentGroupData->pdSteering();
	float4 const*	pdPosition			= m_pAgentGroupData->pdPosition();
	float4 const*	pdDirection			= m_pAgentGroupData->pdDirection();
	float const*	pdSpeed				= m_pAgentGroupData->pdSpeed();
	uint *			pdAppliedKernels	= m_pAgentGroupData->pdAppliedKernels();

	uint const&		numAgents			= getNumAgents();

	SteerForPursueCUDAKernel<<< grid, block >>>(	pdPosition,
													pdDirection,
													pdSpeed,

													m_targetPosition,
													m_targetDirection,
													m_targetVelocity,
													m_targetSpeed,

													pdSteering,

													numAgents,
													m_fMaxPredictionTime,
													m_fWeight,
													pdAppliedKernels,
													m_doNotApplyWith
													);
	cutilCheckMsg( "SteerForPursueCUDAKernel failed." );
	//CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

void SteerForPursueCUDA::close(void)
{
	// Device data has changed. Instruct the AgentGroup it needs to synchronize the host.
	m_pAgentGroup->SetSyncHost();
}
