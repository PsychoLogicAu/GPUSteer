#include "hip/hip_runtime.h"
#include "SteerForPursueCUDA.h"

#include "../AgentGroupData.cuh"

using namespace OpenSteer;

// Kernel function prototype.
extern "C"
{
	__global__ void SteerForPursueCUDAKernel(	float3 * pdSteering, float3 const* pdPosition, float3 const* pdForward, float const* pdSpeed, 
												float3 const targetPosition, float3 const targetForward, float3 const targetVelocity, float const targetSpeed,
												size_t const numAgents, float const maxPredictionTime,
												float const fWeight, uint * pdAppliedKernels, uint const doNotApplyWith
												);
}

SteerForPursueCUDA::SteerForPursueCUDA(		AgentGroup * pAgentGroup, 
											float3 const& targetPosition, float3 const& targetForward, float3 const& targetVelocity, float const& targetSpeed,
											const float fMaxPredictionTime,
											float const fWeight, uint const doNotApplyWith
											)
:	AbstractCUDAKernel( pAgentGroup, fWeight, doNotApplyWith ),
	m_targetPosition( targetPosition ),
	m_targetForward( targetForward ),
	m_targetVelocity( targetVelocity ),
	m_targetSpeed( targetSpeed ),
	m_fMaxPredictionTime( fMaxPredictionTime )
{ }

void SteerForPursueCUDA::init(void)
{ }

void SteerForPursueCUDA::run(void)
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	// Gether the required device pointers.
	float3 *		pdSteering			= m_pAgentGroupData->pdSteering();
	float3 const*	pdPosition			= m_pAgentGroupData->pdPosition();
	float3 const*	pdForward			= m_pAgentGroupData->pdForward();
	float const*	pdSpeed				= m_pAgentGroupData->pdSpeed();
	uint *			pdAppliedKernels	= m_pAgentGroupData->pdAppliedKernels();

	SteerForPursueCUDAKernel<<< grid, block >>>(	pdSteering, pdPosition, pdForward, pdSpeed,
													m_targetPosition, m_targetForward, m_targetVelocity, m_targetSpeed,
													getNumAgents(), m_fMaxPredictionTime, m_fWeight, pdAppliedKernels, m_doNotApplyWith
													);
	cutilCheckMsg( "SteerForPursueCUDAKernel failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

void SteerForPursueCUDA::close(void)
{
	// Device data has changed. Instruct the AgentGroup it needs to synchronize the host.
	m_pAgentGroup->SetSyncHost();
}
