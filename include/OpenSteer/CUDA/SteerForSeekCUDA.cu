#include "hip/hip_runtime.h"
#include "SteerForSeekCUDA.h"

#include "../VehicleGroupData.cuh"

using namespace OpenSteer;

// Kernel function prototype.
extern "C"
{
	__global__ void SteerForSeekCUDAKernel( float3 * pdSteering, float3 const* pdPosition, float3 const* pdForward, float3 const target, size_t const numAgents, float const fWeight );
}

SteerForSeekCUDA::SteerForSeekCUDA( VehicleGroup * pVehicleGroup, float3 const& target, float const fWeight )
:	AbstractCUDAKernel( pVehicleGroup, fWeight ),
	m_target( target )
{
}

void SteerForSeekCUDA::init( void )
{
	// Nothing to do.
}

void SteerForSeekCUDA::run(void)
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	// Gather the required device pointers.
	float3 * pdSteering = m_pVehicleGroupData->pdSteering();
	float3 const* pdPosition = m_pVehicleGroupData->pdPosition();
	float3 const* pdForward = m_pVehicleGroupData->pdForward();

	SteerForSeekCUDAKernel<<< grid, block >>>( pdSteering, pdPosition, pdForward, m_target, getNumAgents(), m_fWeight );
	cutilCheckMsg( "SteerForSeekCUDAKernel failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

void SteerForSeekCUDA::close(void)
{
	// Device data has changed. Instruct the VehicleGroup it needs to synchronize the host.
	m_pVehicleGroup->SetSyncHost();
}
