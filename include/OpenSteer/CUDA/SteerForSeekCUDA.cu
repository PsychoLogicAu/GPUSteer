#include "hip/hip_runtime.h"
#include "SteerForSeekCUDA.cuh"

#include "../AgentGroupData.cuh"

using namespace OpenSteer;

// Kernel function prototype.
extern "C"
{
	__global__ void SteerForSeekCUDAKernel(	float4 *		pdSteering,
											float4 const*	pdPosition,
											float4 const*	pdDirection,

											float3 const	target,

											size_t const	numAgents,
											float const		fWeight,
											uint *			pdAppliedKernels,
											uint const		doNotApplyWith
											);
}

SteerForSeekCUDA::SteerForSeekCUDA( AgentGroup * pAgentGroup, float3 const& target, float const fWeight, uint const doNotApplyWith )
:	AbstractCUDAKernel( pAgentGroup, fWeight, doNotApplyWith ),
	m_target( target )
{
}

void SteerForSeekCUDA::init( void )
{
	// Nothing to do.
}

void SteerForSeekCUDA::run(void)
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	// Gather the required device pointers.
	float4 *		pdSteering			= m_pAgentGroupData->pdSteering();
	float4 const*	pdPosition			= m_pAgentGroupData->pdPosition();
	float4 const*	pdDirection			= m_pAgentGroupData->pdDirection();

	uint *			pdAppliedKernels	= m_pAgentGroupData->pdAppliedKernels();

	uint const&		numAgents			= getNumAgents();

	SteerForSeekCUDAKernel<<< grid, block >>>( pdSteering,
												pdPosition,
												pdDirection,
												
												m_target,
												
												numAgents,
												m_fWeight,
												pdAppliedKernels,
												m_doNotApplyWith
												);
	cutilCheckMsg( "SteerForSeekCUDAKernel failed." );
	//CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

void SteerForSeekCUDA::close(void)
{
	// Device data has changed. Instruct the AgentGroup it needs to synchronize the host.
	m_pAgentGroup->SetSyncHost();
}
