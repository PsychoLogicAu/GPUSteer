#include "hip/hip_runtime.h"
#include "KNNBinningCUDA.cuh"

using namespace OpenSteer;

#include <thrust/sort.h>

#include "DebugUtils.h"

// Kernel file function prototypes.
extern "C"
{
	// Bind texCellIndices to the hipArray.
	__host__ void KNNBinningCUDABindTexture( hipArray * pCudaArray );
	__host__ void KNNBinningCUDAUnbindTexture( void );

	// Kernel to set initial bin indices of vehicles in the simulation.
	__global__ void KNNBinningBuildDB(	float3 const*	pdPosition,				// In:	Positions of each agent.
										size_t *		pdAgentIndices,			// Out:	Indices of each agent.
										size_t *		pdCellIndices,			// Out:	Indices of the cell each agent is in.
										size_t const	numAgents/*,				// In:	Number of agents in the simulation.
										float3 const	worldSize				// In:	Extents of the world (for normalizing the positions).*/
										);

	// Kernel to sort position/direction/speed based on pdAgentIndices, and to compute start and end indices of cells.
	__global__ void KNNBinningReorderData(	float3 const*	pdPosition,			// In: Agent positions.
											float3 const*	pdDirection,		// In: Agent directions.
											float const*	pdSpeed,			// In: Agent speeds.
					
											uint const*		pdAgentIndices,		// In: (sorted) agent index.
											uint const*		pdCellIndices,		// In: (sorted) cell index agent is in.

											float3 *		pdPositionSorted,	// Out: Sorted agent positions.
											float3 *		pdDirectionSorted,	// Out: Sorted agent directions.
											float *			pdSpeedSorted,		// Out: Sorted agent speeds.

											uint *			pdCellStart,		// Out: Start index of this cell in pdCellIndices.
											uint *			pdCellEnd,			// Out: End index of this cell in pdCellIndices.

											size_t const	numAgents
											);

	__global__ void KNNBinningKernel(	float3 const*	pdPositionSorted,	// In:	(sorted) Agent positions.

										uint const*		pdAgentIndices,		// In:	(sorted) Indices of each agent.
										uint const*		pdCellIndices,		// In:	(sorted) Indices of the cell each agent is in.
									
										uint const*		pdCellStart,		// In:	Start index of each cell in pdCellIndices.
										uint const*		pdCellEnd,			// In:	End index of each cell in pdCellIndices.

										uint *			pdKNNIndices,		// Out:	Indices of K Nearest Neighbors in pdPosition.
										float *			pdKNNDistances,		// Out:	Distances of the K Nearest Neighbors in pdPosition.

										size_t const	k,					// In:	Number of neighbors to consider.
										size_t const	radius,				// In:	Maximum radius (in cells) to consider.
										size_t const	numAgents			// In:	Number of agents in the simulation.
										);
}

KNNBinningCUDA::KNNBinningCUDA( VehicleGroup * pVehicleGroup )
:	AbstractCUDAKernel( pVehicleGroup )
{
	m_nCells = m_pVehicleGroup->GetBinData().getNumCells();
	m_pNearestNeighborData = &pVehicleGroup->GetNearestNeighborData();
}

void KNNBinningCUDA::init( void )
{
	// Bind the cell indices texture.
	KNNBinningCUDABindTexture( m_pVehicleGroup->GetBinData().pdCellIndexArray() );

	CUDA_SAFE_CALL( hipMalloc( &m_pdCellStart, m_nCells * sizeof(uint) ) );
	CUDA_SAFE_CALL( hipMalloc( &m_pdCellEnd, m_nCells * sizeof(uint) ) );
}

void KNNBinningCUDA::run( void )
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	size_t const&	numAgents = getNumAgents();
	uint const&		k = m_pNearestNeighborData->k();
	float3 const&	worldSize = m_pVehicleGroup->GetBinData().WorldSize();

	// Gather the required device pointers.
	float3 const*	pdPosition = m_pVehicleGroupData->pdPosition();
	float3 const*	pdDirection = m_pVehicleGroupData->pdForward();
	float const*	pdSpeed = m_pVehicleGroupData->pdSpeed();

	// Pointers to output data.
	uint *			pdKNNIndices = m_pNearestNeighborData->pdKNNIndices();
	float *			pdKNNDistances = m_pNearestNeighborData->pdKNNDistances();

	uint *			pdCellIndices = m_pNearestNeighborData->pdCellIndices();

	uint *			pdCellIndicesSorted = m_pNearestNeighborData->pdCellIndicesSorted();
	uint *			pdAgentIndicesSorted = m_pNearestNeighborData->pdAgentIndicesSorted();

	float3 *		pdPositionSorted = m_pNearestNeighborData->pdPositionSorted();
	float3 *		pdDirectionSorted = m_pNearestNeighborData->pdDirectionSorted();
	float *			pdSpeedSorted = m_pNearestNeighborData->pdSpeedSorted();

	//
	//	TIMING: hard to get exact times with profiling, too many operations.
	//
	// Events for timing the complete operation.
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );

	// Build the database (get the bin indices for the agents).
	KNNBinningBuildDB<<< grid, block >>>( pdPosition, pdAgentIndicesSorted, pdCellIndices, numAgents );
	cutilCheckMsg( "KNNBinningBuildDB failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	// Copy pdCellIndices to pdCellIndicesSorted.
	CUDA_SAFE_CALL( hipMemcpy( pdCellIndicesSorted, pdCellIndices, numAgents * sizeof(uint), hipMemcpyDeviceToDevice ) );

	// Sort m_pAgentIndices on m_pdCellIndicesSorted using thrust.
	thrust::sort_by_key(	thrust::device_ptr<uint>( pdCellIndicesSorted ),
							thrust::device_ptr<uint>( pdCellIndicesSorted + numAgents ),
							thrust::device_ptr<uint>( pdAgentIndicesSorted ) );

	// Set all cells to empty.
	CUDA_SAFE_CALL( hipMemset( m_pdCellStart, 0xffffffff, m_nCells * sizeof(uint) ) );

	KNNBinningReorderData<<< grid, block >>>(	pdPosition, pdDirection, pdSpeed,
												pdAgentIndicesSorted, pdCellIndicesSorted,
												pdPositionSorted, pdDirectionSorted, pdSpeedSorted,
												m_pdCellStart, m_pdCellEnd,
												numAgents
												);
	cutilCheckMsg( "KNNBinningReorderData failed" );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	// Compute the size of shared memory needed for each block.
	size_t shMemSize = k * THREADSPERBLOCK * (sizeof(float) + sizeof(uint));

	KNNBinningKernel<<< grid, block, shMemSize >>>(	pdPositionSorted,
													pdAgentIndicesSorted, pdCellIndicesSorted,
													m_pdCellStart, m_pdCellEnd,
													pdKNNIndices, pdKNNDistances,
													k, 1, numAgents
											);
	cutilCheckMsg( "KNNBinningKernel failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	//
	//	TIMING:
	//
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	
	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );
	char szString[128] = {0};
	sprintf_s( szString, "%f\n", elapsedTime );
	OutputDebugString( szString );


	// Destroy the events.
	hipEventDestroy( start );
	hipEventDestroy( stop );
}

void KNNBinningCUDA::close( void )
{
	// Unbind the texture.
	KNNBinningCUDAUnbindTexture();

	//CUDA_SAFE_CALL( hipFree( m_pdCellIndices ) );
	//CUDA_SAFE_CALL( hipFree( m_pdAgentIndices ) );

	CUDA_SAFE_CALL( hipFree( m_pdCellStart ) );
	CUDA_SAFE_CALL( hipFree( m_pdCellEnd ) );
}