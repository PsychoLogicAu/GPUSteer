#include "hip/hip_runtime.h"
#include "SteerForPursuitCUDA.h"

#include "CUDAKernelGlobals.cuh"

extern "C"
{
	__global__ void SteerToAvoidNeighborsCUDAKernel(	uint const*		pdKNNIndices,			// In:		Indices of the KNN for each agent.
														float const*	pdKNNDistances,			// In:		Distances to the KNN for each agent.
														size_t const	k,						// In:		Number of KNN for each agent.

														float3 const*	pdPosition,				// In:		Positions of each agent.
														float3 const*	pdDirection,			// In:		Directions of facing for each agent.
														float const*	pdRadius,				// In:		Radius of each agent.
														float3 const*	pdSide,					// In:		Side direction for each agent.

														float *			pdSpeed,				// In/Out:	Speed of each agent.
														float3 *		pdSteering,				// Out:		Steering vectors for each agent.

														float const		minTimeToCollision,		// In:		Look-ahead time for collision avoidance.
														float const		minSeparationDistance,	// In:		Distance to consider 'close' neighbors.

														size_t const	numAgents
														);

	__global__ void SteerToAvoidCloseNeighborsCUDAKernel(	uint const*		pdKNNIndices,
		float const*	pdKNNDistances,
		size_t const	k,

		float3 const*	pdPosition,
		float3 const*	pdDirection,
		float const*	pdRadius,

		float3 *		pdSteering,
		float *			pdSpeed,

		float const		minSeparationDistance,

		size_t const	numAgents
		);
}

// DEPRECATED: this has been handled within SteerToAvoidNeighborsCUDAKernel
__global__ void SteerToAvoidCloseNeighborsCUDAKernel(	uint const*		pdKNNIndices,
													 float const*	pdKNNDistances,
													 size_t const	k,

													 float3 const*	pdPosition,
													 float3 const*	pdDirection,
													 float const*	pdRadius,
													 float *			pdSpeed,

													 float3 *		pdSteering,

													 float const		minSeparationDistance,

													 size_t const	numAgents
														)
{
	int const index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( index >= numAgents )
		return;

	extern __shared__ float shKNNDistances[];
	uint *	shKNNIndices = (uint*)shKNNDistances + (THREADSPERBLOCK*k);

	__shared__ float3 shPosition[THREADSPERBLOCK];
	__shared__ float3 shDirection[THREADSPERBLOCK];
	__shared__ float3 shSteering[THREADSPERBLOCK];
	__shared__ float shRadius[THREADSPERBLOCK];
	__shared__ float shSpeed[THREADSPERBLOCK];

	// Copy required global memory to shared.
	FLOAT3_GLOBAL_READ( shPosition, pdPosition );
	FLOAT3_GLOBAL_READ( shDirection, pdDirection );
	FLOAT3_GLOBAL_READ( shSteering, pdSteering );
	RADIUS_SH( threadIdx.x ) = RADIUS( index );
	__syncthreads();
	SPEED_SH( threadIdx.x ) = SPEED( index );
	__syncthreads();
	for( uint i = 0; i < k; i++ )
	{
		shKNNDistances[threadIdx.x*k + i] = pdKNNDistances[index*k + i];
		shKNNIndices[threadIdx.x*k + i] = pdKNNIndices[index*k + i];
	}
	__syncthreads();

	uint threatIndex;

	// For each of the KNN of this agent...
	for( int i = 0; i < k; i++ )
	{
		threatIndex = shKNNIndices[(threadIdx.x * k) + i];

		if( threatIndex >= numAgents )
			break;

		float const sumOfRadii = RADIUS_SH( threadIdx.x ) + RADIUS( threatIndex );
		float const minCenterToCenter = minSeparationDistance + sumOfRadii;
		float3 const offset = float3_subtract( POSITION( threatIndex ), POSITION_SH( threadIdx.x ) );
		//float const currentDistance = float3_length( offset );

		// Distance was computed in KNN step. Don't waste time :)
		float const currentDistance = shKNNDistances[(threadIdx.x * k) + i];

		//if( currentDistance < sumOfRadii )
		//{
		//	// Agents are interpenetrating. Bad!

		//	// If the agent at threatIndex is ahead of me...
		//	if( float3_dot( DIRECTION_SH( threadIdx.x ), offset ) > 0.f )
		//	{
		//		// I should slow down.
		//		SPEED_SH( threadIdx.x ) *= (currentDistance / minSeparationDistance);
		//	}
		//}

		if( currentDistance < minCenterToCenter )	// Other agent is within critical range.
		{
			// Steer hard to dodge the other agent.
			STEERING_SH( threadIdx.x ) = float3_perpendicularComponent( float3_minus( offset ), DIRECTION_SH( threadIdx.x ) );

			// TESTING: slow down if collision iminent
			// If the agent at threatIndex is ahead of me...
			if( float3_dot( DIRECTION_SH( threadIdx.x ), offset ) > 0.f )
			{
				// I should slow down.
				SPEED_SH( threadIdx.x ) *= (currentDistance / minCenterToCenter);
			}
		}
	}

	__syncthreads();

	// Write the steering vectors back to global memory.
	FLOAT3_GLOBAL_WRITE( pdSteering, shSteering );

	SPEED( index ) = SPEED_SH( threadIdx.x );
}

// Given the time until nearest approach (predictNearestApproachTime)
// determine position of each agent at that time, and the distance
// between them.
__inline__ __device__ float computeNearestApproachPositions( float3 const& position, float3 const& direction, float const& speed, float3 const& otherPosition, float3 const& otherDirection, float const& otherSpeed, float const& time, float3 & threatNearestPosition, float3 & myNearestPosition )
{
	float3 const myTravel =		float3_scalar_multiply( direction, speed * time );
	float3 const otherTravel =	float3_scalar_multiply( otherDirection, otherSpeed * time );

	myNearestPosition =			float3_add( position, myTravel );
	threatNearestPosition =		float3_add( otherPosition, otherTravel );

	return float3_distance( myNearestPosition, threatNearestPosition );
}

// Given two agents, based on their current positions and velocities,
// determine the time until nearest approach.
__inline__ __device__ float predictNearestApproachTime( float3 const& position, float3 const& direction, float const& speed, float3 const& otherPosition, float3 const& otherDirection, float const& otherSpeed )
{
	// imagine we are at the origin with no velocity,
	// compute the relative velocity of the other vehicle
	float3 const myVelocity = float3_scalar_multiply( direction, speed );
	float3 otherVelocity = float3_scalar_multiply( otherDirection, otherSpeed );
	float3 const relVelocity = float3_subtract( otherVelocity, myVelocity );
	float const relSpeed = float3_length( relVelocity );

	// for parallel paths, the vehicles will always be at the same distance,
	// so return 0 (aka "now") since "there is no time like the present"
	if( relSpeed == 0.f )
		return 0.f;

	// Now consider the path of the other vehicle in this relative
	// space, a line defined by the relative position and velocity.
	// The distance from the origin (our vehicle) to that line is
	// the nearest approach.

	// Take the unit tangent along the other vehicle's path
	float3 const relTangent = float3_scalar_divide( relVelocity, relSpeed );

	// find distance from its path to origin (compute offset from
	// other to us, find length of projection onto path)
	float3 const relPosition = float3_subtract( position, otherPosition );
	float const projection = float3_dot( relTangent, relPosition );

	return projection / relSpeed;
}

__global__ void SteerToAvoidNeighborsCUDAKernel(	uint const*		pdKNNIndices,			// In:		Indices of the KNN for each agent.
													float const*	pdKNNDistances,			// In:		Distances to the KNN for each agent.
													size_t const	k,						// In:		Number of KNN for each agent.

													float3 const*	pdPosition,				// In:		Positions of each agent.
													float3 const*	pdDirection,			// In:		Directions of facing for each agent.
													float const*	pdRadius,				// In:		Radius of each agent.
													float3 const*	pdSide,					// In:		Side direction for each agent.

													float *			pdSpeed,				// In/Out:	Speed of each agent.
													float3 *		pdSteering,				// Out:		Steering vectors for each agent.

													float const		minTimeToCollision,		// In:		Look-ahead time for collision avoidance.
													float const		minSeparationDistance,	// In:		Distance to consider 'close' neighbors.

													size_t const	numAgents
												)
{
	int const index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( index >= numAgents )
		return;

	extern __shared__ float shKNNDistances[];
	uint *	shKNNIndices = (uint*)shKNNDistances + (THREADSPERBLOCK*k);

	__shared__ float3 shPosition[THREADSPERBLOCK];
	__shared__ float3 shDirection[THREADSPERBLOCK];
	__shared__ float3 shSide[THREADSPERBLOCK];
	__shared__ float shSpeed[THREADSPERBLOCK];
	__shared__ float shRadius[THREADSPERBLOCK];
	__shared__ float3 shSteering[THREADSPERBLOCK];

	// Load this block's data into shared memory.
	FLOAT3_GLOBAL_READ( shPosition, pdPosition );
	FLOAT3_GLOBAL_READ( shDirection, pdDirection );
	FLOAT3_GLOBAL_READ( shSide, pdSide );
	FLOAT3_GLOBAL_READ( shSteering, pdSteering );
	SPEED_SH( threadIdx.x ) = SPEED( index );
	__syncthreads();
	RADIUS_SH( threadIdx.x ) = RADIUS( index );
	__syncthreads();
	// Load the KNN data from global memory.
	for( uint i = 0; i < k; i++ )
	{
		shKNNIndices[threadIdx.x*k + i] = pdKNNIndices[index*k + i];
		shKNNDistances[threadIdx.x*k + i] = pdKNNDistances[index*k + i];
	}
	__syncthreads();

	// If there is a steering vector set, it was done by SteerToAvoidCloseNeighbors. In that case, we should do nothing here.
	if( ! float3_equals( STEERING_SH( threadIdx.x ), float3_zero() ) )
		return;

	// Find the agent which is closest to collision
	float minTime = minTimeToCollision;
	float steer = 0.f;

	uint threatIndex = UINT_MAX;				// Index of the nearest threat.
	float threatDistance = FLT_MAX;				// Distance of the nearest threat.

	float3 threatPositionAtNearestApproach;
	float3 myPositionAtNearestApproach;

	// For each of the neighboring vehicles, determine which (if any)
	// pose the most immediate threat of collision.
	uint otherIndex;
	float otherDistance;
	for( uint i = 0; i < k; i++ )
	{
		otherIndex = shKNNIndices[threadIdx.x * k + i];
		otherDistance = shKNNDistances[threadIdx.x * k + i];

		// Check for end of KNN (will be UINT_MAX if there are no more).
		if( otherIndex >= numAgents )
			break;

		// avoid when future positions are this close (or less)
		float const sumOfRadii = RADIUS_SH( threadIdx.x ) + RADIUS( otherIndex );

		// Check for a 'close' neighbor.
		if( otherDistance < (minSeparationDistance + sumOfRadii) && otherDistance < threatDistance )
		{
			minTime = 0;
			threatIndex = otherIndex;
			threatDistance = otherDistance;
			break;
		}

		// predicted time until nearest approach of "this" and "other"
		float const time = predictNearestApproachTime(	POSITION_SH( threadIdx.x ), DIRECTION_SH( threadIdx.x ), SPEED_SH( threadIdx.x ),
														POSITION( otherIndex ), DIRECTION( otherIndex ), SPEED( otherIndex )
														);

		// If the time is in the future, sooner than any other threatened collision...
		if(	time >= 0		&&	// Time is in the future.
			time < minTime )	// Sooner than other threats.
		{
			// if the two will be close enough to collide, make a note of it
			if( computeNearestApproachPositions(	POSITION_SH( threadIdx.x ), DIRECTION_SH( threadIdx.x ), SPEED_SH( threadIdx.x ),
													POSITION( otherIndex ), DIRECTION( otherIndex ), SPEED( otherIndex ),
													time,
													threatPositionAtNearestApproach, myPositionAtNearestApproach
													)
				<
				sumOfRadii )
			{
				minTime = time;
				threatIndex = otherIndex;
				threatDistance = otherDistance;
			}
		}
	}

	// Was there a 'threat' found?
	if( UINT_MAX != threatIndex )
	{
		// FIXME: already done
		float const sumOfRadii = RADIUS_SH( threadIdx.x ) + RADIUS( threatIndex );
		float const minCenterToCenter = minSeparationDistance + sumOfRadii;
		float3 const offset = float3_subtract( POSITION( threatIndex ), POSITION_SH( threadIdx.x ) );

		// parallel: +1, perpendicular: 0, anti-parallel: -1
		float const parallelness = float3_dot( DIRECTION_SH( threadIdx.x ), DIRECTION( threatIndex ) );
		float const angle = 0.707f;

		if( threatDistance < minCenterToCenter )	// Other agent is within 'close' range.
		{
			// Steer hard to dodge the other agent.
			STEERING_SH( threadIdx.x ) = float3_perpendicularComponent( float3_minus( offset ), DIRECTION_SH( threadIdx.x ) );

			// Slow down if collision iminent
			// If the agent at threatIndex is ahead of me...
			if(	float3_dot( DIRECTION_SH( threadIdx.x ), offset ) > 0.f &&		// Other agent is in front.
				SPEED_SH( threadIdx.x ) > SPEED( threatIndex )					// Moving faster than the threat agent.
				)
			{
				// I should slow down.
				SPEED_SH( threadIdx.x ) *= (threatDistance / minCenterToCenter);
			}
		}
		else
		{


			if( parallelness < -angle )		// anti-parallel "head on" paths:
			{
				// steer away from future threat position
				float3 offset = float3_subtract( threatPositionAtNearestApproach, POSITION_SH( threadIdx.x ) );
				float sideDot = float3_dot( offset, SIDE_SH( threadIdx.x ) );
				steer = (sideDot > 0) ? -1.0f : 1.0f;
			}
			else
			{
				if (parallelness > angle)	// parallel paths: steer away from threat
				{

					float3 offset = float3_subtract( POSITION( threatIndex ), POSITION_SH( threadIdx.x ) );
					float sideDot = float3_dot( offset, SIDE_SH( threadIdx.x ) );
					steer = (sideDot > 0) ? -1.0f : 1.0f;
				}
				else						// perpendicular paths: steer behind threat
				{
					// (only the slower of the two does this)
					if( SPEED( threatIndex ) <= SPEED_SH( threadIdx.x ) )
					{
						float sideDot = float3_dot( SIDE_SH( threadIdx.x ), float3_scalar_multiply( DIRECTION( threatIndex ), SPEED( threatIndex ) ) );
						steer = (sideDot > 0) ? -1.0f : 1.0f;
					}
				}
			}
		}
	}

	STEERING_SH( threadIdx.x ) = float3_scalar_multiply( SIDE_SH( threadIdx.x ), steer );

	__syncthreads();

	// Write the steering vectors and speeds to global memory.
	FLOAT3_GLOBAL_WRITE( pdSteering, shSteering );
	SPEED( index ) = SPEED_SH( threadIdx.x );
}
