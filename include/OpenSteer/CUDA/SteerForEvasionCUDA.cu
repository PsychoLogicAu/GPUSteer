#include "hip/hip_runtime.h"
#include "SteerForEvasionCUDA.cuh"

extern "C"
{
	__global__ void SteerForEvasionKernel(	// Agent data.
											float4 const*	pdPosition,
											float4 const*	pdDirection,
											float4 *		pdSteering,

											float3 const	menacePosition,
											float3 const	menaceDirection,
											float const		menaceSpeed,
											
											float const		maxPredictionTime,

											size_t const	numAgents,

											float const		fWeight,
											uint *			pdAppliedKernels,
											uint const		doNotApplyWith
										  );
}

using namespace OpenSteer;

SteerForEvadeCUDA::SteerForEvadeCUDA( AgentGroup * pAgentGroup, float3 const& menacePosition, float3 const& menaceDirection, float const menaceSpeed, float const fMaxPredictionTime, float const fWeight, uint const doNotApplyWith )
:	AbstractCUDAKernel( pAgentGroup, fWeight, doNotApplyWith ),
	m_fMaxPredictionTime( fMaxPredictionTime ),
	m_menacePosition( menacePosition ),
	m_menaceDirection( menaceDirection ),
	m_menaceSpeed( menaceSpeed )
{
	// Nothing to do.
}

void SteerForEvadeCUDA::init( void )
{
	// Nothing to do.
}

void SteerForEvadeCUDA::run( void )
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	// Gather required device data.
	float4 const*	pdPosition			= m_pAgentGroupData->pdPosition();
	float4 const*	pdDirection			= m_pAgentGroupData->pdDirection();
	float4 *		pdSteering			= m_pAgentGroupData->pdSteering();

	uint const		numAgents			= getNumAgents();

	uint *			pdAppliedKernels	= m_pAgentGroupData->pdAppliedKernels();

	SteerForEvasionKernel<<< grid, block >>>(	// Agent data.
												pdPosition,
												pdDirection,
												pdSteering,

												m_menacePosition,
												m_menaceDirection,
												m_menaceSpeed,

												m_fMaxPredictionTime,

												numAgents,
												m_fWeight,
												pdAppliedKernels,
												m_doNotApplyWith
												);
	cutilCheckMsg( "SteerForEvasionKernel failed." );
	//CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

void SteerForEvadeCUDA::close( void )
{
	// Agent group data may have changed.
	m_pAgentGroup->SetSyncHost();
}
