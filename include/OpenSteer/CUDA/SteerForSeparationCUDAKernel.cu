#include "hip/hip_runtime.h"
#include "SteerForSeparationCUDA.cuh"

#include "CUDAKernelGlobals.cuh"

#include "FlockingCommon.cuh"

extern "C"
{
	__host__ void SteerForSeparationKernelBindTextures(	float4 const*	pdBPosition,
														uint const		numB
														);
	__host__ void SteerForSeparationKernelUnindTextures( void );

	__global__ void SteerForSeparationKernel(	float4 const*	pdPosition,
												float4 const*	pdDirection,
												float4 *		pdSteering,
												size_t const	numA,

												uint const*		pdKNNIndices,
												size_t const	k,

												uint const		numB,

												float const		minDistance,
												float const		maxDistance,
												float const		cosMaxAngle,

												float const		fWeight,
												uint *			pdAppliedKernels,
												uint const		doNotApplyWith
												);
}

texture< float4, hipTextureType1D, hipReadModeElementType>	texBPosition;

__host__ void SteerForSeparationKernelBindTextures(	float4 const*	pdBPosition,
													uint const		numB
													)
{
	static hipChannelFormatDesc const float4ChannelDesc = hipCreateChannelDesc< float4 >();

	CUDA_SAFE_CALL( hipBindTexture( NULL, texBPosition, pdBPosition, float4ChannelDesc, numB * sizeof(float4) ) );
}

__host__ void SteerForSeparationKernelUnindTextures( void )
{
	CUDA_SAFE_CALL( hipUnbindTexture( texBPosition ) );
}

__global__ void SteerForSeparationKernel(	float4 const*	pdPosition,
											float4 const*	pdDirection,
											float4 *		pdSteering,
											size_t const	numA,

											uint const*		pdKNNIndices,
											size_t const	k,

											uint const		numB,

											float const		minDistance,
											float const		maxDistance,
											float const		cosMaxAngle,

											float const		fWeight,
											uint *			pdAppliedKernels,
											uint const		doNotApplyWith
											)
{
	uint const index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( index >= numA )
		return;

	if( pdAppliedKernels[ index ] & doNotApplyWith )
		return;

	extern __shared__ uint shKNNIndices[];

	__shared__ float3	shSteering[THREADSPERBLOCK];
	__shared__ float3	shPosition[THREADSPERBLOCK];
	__shared__ float3	shDirection[THREADSPERBLOCK];

	// Copy required from global memory.
	STEERING_SH( threadIdx.x )	= STEERING_F3( index );
	POSITION_SH( threadIdx.x )	= POSITION_F3( index );
	DIRECTION_SH( threadIdx.x )	= DIRECTION_F3( index );

	for( int i = 0; i < k; i++ )
		shKNNIndices[threadIdx.x*k + i] = pdKNNIndices[index*k + i];
	__syncthreads();

    // steering accumulator and count of neighbors, both initially zero
	float3 steering = { 0.f, 0.f, 0.f };
    uint neighbors = 0;

    // For each agent in this agent's KNN neighborhood...
	for( uint i = 0; i < k; i++ )
	{
		uint BIndex = shKNNIndices[threadIdx.x * k + i];

		// Check for end of KNN.
		if( BIndex >= numB )
			break;

		float3 const bPosition = make_float3( tex1Dfetch( texBPosition, BIndex ) );

		if( inBoidNeighborhood( POSITION_SH( threadIdx.x ), DIRECTION_SH( threadIdx.x ), bPosition, minDistance, maxDistance, cosMaxAngle ) )
		{
			float3 const offset = float3_subtract( bPosition, POSITION_SH( threadIdx.x ) );
			float const distanceSquared = float3_dot( offset, offset );
			steering = float3_add( steering, float3_scalar_divide( offset, -distanceSquared ) );

			neighbors++;
		}
	}

    // divide by neighbors, then normalize to pure direction
	if( neighbors > 0 )
		steering = float3_normalize( float3_scalar_divide( steering, (float)neighbors ) );

	// Apply the weight.
	steering = float3_scalar_multiply( steering, fWeight );

	// Set the applied kernel bit.
	if( ! float3_equals( steering, float3_zero() ) )
		pdAppliedKernels[ index ] |= KERNEL_SEPARATION_BIT;

	// Add into the steering vector.
	STEERING_SH( threadIdx.x ) = float3_add( steering, STEERING_SH( threadIdx.x ) );

	// Write back to global memory.
	STEERING( index ) = STEERING_SH_F4( threadIdx.x );
}
