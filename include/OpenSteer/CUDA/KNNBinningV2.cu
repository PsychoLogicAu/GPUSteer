#include "hip/hip_runtime.h"
#include "KNNBinningV2.cuh"

using namespace OpenSteer;

#include "KNNBinData.cuh"

#include <thrust/sort.h>

#define TIMING

// Kernel file function prototypes.
extern "C"
{
	// Bind texCellIndices to the hipArray.
	__host__ void KNNBinningV2BindTexture( hipArray * pCudaArray );
	__host__ void KNNBinningV2UnbindTexture( void );

	__host__ void KNNBinningV2KernelBindTextures(			uint const*		pdBCellStart,
														uint const*		pdBCellEnd,
														uint const*		pdBIndices,
														float4 const*	pdBPositionSorted,
														uint const*		pdCellNeighbors,
														uint const		numCells,
														uint const		numB,
														uint const		neighborsPerCell
														);

	__host__ void KNNBinningV2KernelUnbindTextures( void );
	__host__ void KNNBinningV2ReorderDBBindTextures(	float4 const*	pdPosition,
													uint const		numAgents );
	__host__ void KNNBinningV2ReorderDBUnbindTextures( void );

	__global__ void KNNBinningV2BuildDB(					float4 const*	pdPosition,				// In:	Positions of each agent.
														size_t *		pdAgentIndices,			// Out:	Indices of each agent.
														size_t *		pdCellIndices,			// Out:	Indices of the cell each agent is in.
														size_t const	numAgents
														);

	__global__ void KNNBinningV2ReorderDB(				uint const*		pdAgentIndices,		// In: (sorted) agent index.
														uint const*		pdCellIndices,		// In: (sorted) cell index agent is in.

														float4 *		pdPositionSorted,	// Out: Sorted agent positions.

														uint *			pdCellStart,		// Out: Start index of this cell in pdCellIndices.
														uint *			pdCellEnd,			// Out: End index of this cell in pdCellIndices.

														size_t const	numAgents
														);

	__global__ void KNNBinningV2Kernel(					// Group A
														float4 const*	pdAPositionSorted,			// In:	Sorted group A positions.

														uint const*		pdAIndices,					// In:	Sorted group A indices
														uint const*		pdACellIndices,				// In:	Sorted group A cell indices.

														// Cell neighbor info.
														uint const		neighborsPerCell,			// In:	Number of neighbors per cell in the pdCellNeighbors array.
														uint const		radius,						// In:	Search radius (in cells) to consider.

														// Output data.
														uint *			pdKNNIndices,				// Out:	Indices of K Nearest Neighbors in pdPosition.
														float *			pdKNNDistances,				// Out:	Distances of the K Nearest Neighbors in pdPosition.

														uint const		k,							// In:	Number of neighbors to consider.
														uint const		numA,						// In:	Size of group A.
														uint const		numB,						// In:	Size of group B.
														bool const		groupWithSelf				// In:	Are we testing this group with itself? (group A == group B)
														);
}

#pragma region KNNBinningV2UpdateDBCUDA

KNNBinningV2UpdateDBCUDA::KNNBinningV2UpdateDBCUDA( BaseGroup * pGroup, KNNBinDataV2 * pKNNBinData )
:	AbstractCUDAKernel( NULL, 1.f, 0 ),
	m_pGroup( pGroup ),
	m_pKNNBinData( pKNNBinData )
{
	// Nothing to do.
}

void KNNBinningV2UpdateDBCUDA::init( void )
{
	// Bind the lookup texture.
	KNNBinningV2BindTexture( m_pKNNBinData->pdCellIndexArray() );
}

void KNNBinningV2UpdateDBCUDA::run( void )
{
	dim3 grid	= dim3( (m_pGroup->Size() + THREADSPERBLOCK - 1) / THREADSPERBLOCK );
	dim3 block	= dim3( THREADSPERBLOCK );

	// Gather required data.
	float4 const*	pdPosition				= m_pGroup->pdPosition();
	
	uint *			pdCellIndices			= m_pGroup->GetKNNDatabase().pdCellIndices();

	uint *			pdAgentIndicesSorted	= m_pGroup->GetKNNDatabase().pdAgentIndicesSorted();
	uint *			pdCellIndicesSorted		= m_pGroup->GetKNNDatabase().pdCellIndicesSorted();

	float4 *		pdPositionSorted		= m_pGroup->GetKNNDatabase().pdPositionSorted();
	uint *			pdCellStart				= m_pGroup->GetKNNDatabase().pdCellStart();
	uint *			pdCellEnd				= m_pGroup->GetKNNDatabase().pdCellEnd();

	uint const&		numAgents				= m_pGroup->Size();

#if defined TIMING
	//
	//	TIMING: hard to get exact times with profiling, too many operations.
	//
	// Events for timing the complete operation.
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );
#endif

	// Call KNNBinningBuildDB to build the database. 
	KNNBinningV2BuildDB<<< grid, block >>>( pdPosition, pdAgentIndicesSorted, pdCellIndices, numAgents );
	cutilCheckMsg( "KNNBinningV2BuildDB failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	// Copy pdCellIndices to  pdCellIndicesSorted.
	CUDA_SAFE_CALL( hipMemcpy( pdCellIndicesSorted, pdCellIndices, numAgents * sizeof(uint), hipMemcpyDeviceToDevice ) );

	// Sort pdAgentIndicesSorted on pdCellIndicesSorted using thrust.
	thrust::sort_by_key(	thrust::device_ptr<uint>( pdCellIndicesSorted ),
							thrust::device_ptr<uint>( pdCellIndicesSorted + numAgents ),
							thrust::device_ptr<uint>( pdAgentIndicesSorted ) );

	// Set all cells to empty.
	CUDA_SAFE_CALL( hipMemset( pdCellStart, 0xffffffff, m_pKNNBinData->getNumCells() * sizeof(uint) ) );

	// Bind the textures.
	KNNBinningV2ReorderDBBindTextures( pdPosition, numAgents );

	// Call KNNBinningReorderDB to re-order the data in the DB.
	KNNBinningV2ReorderDB<<< grid, block >>>( pdAgentIndicesSorted, pdCellIndicesSorted, pdPositionSorted, pdCellStart, pdCellEnd, numAgents );	cutilCheckMsg( "KNNBinningReorderDB failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	// Unbind the textures.
	KNNBinningV2ReorderDBUnbindTextures();

#if defined TIMING
	//
	//	TIMING:
	//
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	
	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );
	char szString[128] = {0};
	sprintf_s( szString, "KNNBinningV2UpdateDBCUDA,%f\n", elapsedTime );
	//OutputDebugStringToFile( szString );
	OutputDebugString( szString );

	// Destroy the events.
	hipEventDestroy( start );
	hipEventDestroy( stop );
#endif
}

void KNNBinningV2UpdateDBCUDA::close( void )
{
	// Unbind the texture.
	KNNBinningV2UnbindTexture();

	// The AgentGroup's database has now changed.
	m_pGroup->SetSyncHost();
}
#pragma endregion


#pragma region KNNBinningV2

KNNBinningV2CUDA::KNNBinningV2CUDA( AgentGroup * pAgentGroup, KNNData * pKNNData, KNNBinDataV2 * pKNNBinData, BaseGroup * pOtherGroup, uint const searchRadius )
:	AbstractCUDAKernel( pAgentGroup, 1.f, 0 ),
	m_pKNNData( pKNNData ),
	m_pKNNBinData( pKNNBinData ),
	m_pOtherGroup( pOtherGroup ),
	m_searchRadius( searchRadius )
{
}

void KNNBinningV2CUDA::init( void )
{
	// Bind the cell indices texture.
	//KNNBinningV2CUDABindTexture( m_pKNNBinData->pdCellIndexArray() );
}

void KNNBinningV2CUDA::run( void )
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	// Gather the required data.
	float4 const*		pdAPositionSorted		= m_pAgentGroup->GetKNNDatabase().pdPositionSorted();
	uint const*			pdAIndices				= m_pAgentGroup->GetKNNDatabase().pdAgentIndicesSorted();
	uint const*			pdACellIndices			= m_pAgentGroup->GetKNNDatabase().pdCellIndicesSorted();

	float4 const*		pdBPositionSorted		= m_pOtherGroup->GetKNNDatabase().pdPositionSorted();
	uint const*			pdBIndices				= m_pOtherGroup->GetKNNDatabase().pdAgentIndicesSorted();
	uint const*			pdBCellIndices			= m_pOtherGroup->GetKNNDatabase().pdCellIndicesSorted();

	uint const*			pdBCellStart			= m_pOtherGroup->GetKNNDatabase().pdCellStart();
	uint const*			pdBCellEnd				= m_pOtherGroup->GetKNNDatabase().pdCellEnd();

	uint const*			pdCellNeighbors			= m_pKNNBinData->pdCellNeighbors();
	uint const&			neighborsPerCell		= m_pKNNBinData->neighborsPerCell();
	//uint const			neighborsPerCell		= ipow( (m_searchRadius * 2 + 1), (m_pKNNBinData->is3D() ? 3 : 2) );
	uint const&			numCells				= m_pKNNBinData->getNumCells();

	uint *				pdKNNIndices			= m_pKNNData->pdKNNIndices();
	float *				pdKNNDistances			= m_pKNNData->pdKNNDistances();

	uint const&			k						= m_pOtherGroup->GetKNNDatabase().k();
	uint const&			numA					= getNumAgents();
	uint const&			numB					= m_pOtherGroup->Size();

	bool const			groupWithSelf			= m_pAgentGroup == m_pOtherGroup;

	// Compute the size of shared memory needed for each block.
	size_t shMemSize = k * THREADSPERBLOCK * (sizeof(float) + sizeof(uint));

#if defined TIMING
	//
	//	TIMING: hard to get exact times with profiling, too many operations.
	//
	// Events for timing the complete operation.
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );
#endif

	// Bind the textures.
	KNNBinningV2KernelBindTextures(	pdBCellStart, pdBCellEnd, pdBIndices, pdBPositionSorted, pdCellNeighbors, numCells, numB, neighborsPerCell );

	// Call the KNNBinning kernel.
	KNNBinningV2Kernel<<< grid, block, shMemSize >>>(	pdAPositionSorted,
													pdAIndices, pdACellIndices, 
													neighborsPerCell, m_searchRadius,
													pdKNNIndices, pdKNNDistances, k,
													numA, numB, groupWithSelf
													);
	cutilCheckMsg( "KNNBinningV2Kernel failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	// Unbind the textures.
	KNNBinningV2KernelUnbindTextures();

#if defined TIMING
	//
	//	TIMING:
	//
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	
	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );
	char szString[128] = {0};
	sprintf_s( szString, "KNNBinningV2CUDA,%f\n", elapsedTime );
	//OutputDebugStringToFile( szString );
	OutputDebugString( szString );

	// Destroy the events.
	hipEventDestroy( start );
	hipEventDestroy( stop );
#endif
}

void KNNBinningV2CUDA::close( void )
{
	// The KNNData has most likely changed.
	m_pKNNData->setSyncHost();
}

#pragma endregion
