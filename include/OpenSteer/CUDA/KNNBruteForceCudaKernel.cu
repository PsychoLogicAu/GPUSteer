#include "hip/hip_runtime.h"
#include "KNNBruteForceCUDA.cuh"

//#include "../VehicleGroupData.cuh"
#include "../VectorUtils.cuh"

#include "CUDAKernelGlobals.cuh"

// For FLT_MAX.
#include "float.h"

using namespace OpenSteer;

extern "C"
{
	// O(N2) time and memory approach.
	__global__ void KNNBruteForceCUDAKernel(	float3 const*	pdPosition,			// Agent positions.
												float *			pdDistanceMatrix,	// Global storage for distance matrix.
												size_t *		pdIndexMatrix,		// The indices which match postions in pdDistanceMatrix.
												//size_t *		pdKNNIndices,		// Output, indices of K Nearest Neighbors in pdPosition.
												size_t const	k,					// Number of neighbors to consider.
												size_t const	numAgents			// Number of agents in the simulation.
											);

	__global__ void KNNBruteForceCUDAKernelV2(	float3 const*	pdPosition,			// Agent positions.
												uint *			pdKNNIndices,		// Output, indices of K Nearest Neighbors in pdPosition.
												size_t const	k,					// Number of neighbors to consider.
												size_t const	numAgents			// Number of agents in the simulation.
											);

	__global__ void KNNBruteForceCUDAKernelV3(	float3 const*	pdPosition,			// Agent positions.
												uint *			pdKNNIndices,		// Output, indices of K Nearest Neighbors in pdPosition.
												float *			pdKNNDistances,		// Output, distances of the K Nearest Neighbors in pdPosition.
												size_t const	k,					// Number of neighbors to consider.
												size_t const	numAgents,			// Number of agents in the simulation.
												bool const		bSeed = false
											);
	
}

__global__ void KNNBruteForceCUDAKernelV3(	float3 const*	pdPosition,			// Agent positions.
											uint *			pdKNNIndices,		// Output, indices of K Nearest Neighbors in pdPosition.
											float *			pdKNNDistances,		// Output, distances of the K Nearest Neighbors in pdPosition.
											size_t const	k,					// Number of neighbors to consider.
											size_t const	numAgents,			// Number of agents in the simulation.
											bool const		bSeed
										)
{
	int offset = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( offset >= numAgents )
		return;

	// Shared memory for local priority queue computations.
	extern __shared__ float shDist[];				// First half will be treated as the distance values.
	uint * shInd = (uint*)shDist + blockDim.x * k;	// Second half will be treated as the index values.
	
	// Store this thread's agent position in registers.  // TODO: page this read for the block and coalesce.
	float3 position = POSITION( offset );

	// Set all elements of shDist to FLT_MAX.
	for( uint i = 0; i < k; i++ )
		shDist[(threadIdx.x * k) + i] = FLT_MAX;

	__syncthreads();
	
	if( bSeed )
	{
		// Set the seeding values from the previous update.
		for( uint i = 0; i < k; i++ )
		{
			// Get the index of the ith closest agent from the last frame.
			uint const ind = pdKNNIndices[(blockIdx.x * blockDim.x) + (threadIdx.x * k) + i];

			// Compute the distance between this agent and the one at index.
			float const dist = float3_distance( position, POSITION( ind ) );

			if( dist < shDist[(threadIdx.x * k) + (k - 1)] )	// Distance of the kth closest agent.
			{
				// Agent at index i is the new kth closest. Set the distance and index in shared mem.
				shDist[(threadIdx.x * k) + (k - 1)] = dist;
				shInd[(threadIdx.x * k) + (k - 1)] = ind;

				// Bubble the values up... this is necessary as their positions may have changed relative to each other since the last update.
				for( int slot = k - 2; slot >= 0; slot-- )
				{
					if( shDist[(threadIdx.x * k) + slot] > shDist[(threadIdx.x * k) + (slot + 1)] )
					{
						swap( shDist[(threadIdx.x * k) + slot], shDist[(threadIdx.x * k) + (slot + 1)] );
						swap( shInd[(threadIdx.x * k) + slot], shInd[(threadIdx.x * k) + (slot + 1)] );
					}
				}
			}
		}
	}

	__syncthreads();

	// For each of the agents...
	for( uint i = 0; i < numAgents; i++ )
	{
		// Test this... will likely be slower than computing k+1 and discarding the shortest...
		if( i == offset )
			continue;

		// Compute the distance between this agent and the one at i.
		float const dist = float3_distance( position, POSITION( i ) );

		if( dist < shDist[(threadIdx.x * k) + (k - 1)] )	// Distance of the kth closest agent.
		{
			// Agent at index i is the new (at least) kth closest. Set the distance and index in shared mem.
			shDist[(threadIdx.x * k) + (k - 1)] = dist;
			shInd[(threadIdx.x * k) + (k - 1)] = i;

			// Bubble the values up...
			for( int slot = k - 2; slot >= 0; slot-- )
			{
				if( shDist[(threadIdx.x * k) + slot] > shDist[(threadIdx.x * k) + (slot + 1)] )
				{
					swap( shDist[(threadIdx.x * k) + slot], shDist[(threadIdx.x * k) + (slot + 1)] );
					swap( shInd[(threadIdx.x * k) + slot], shInd[(threadIdx.x * k) + (slot + 1)] );
				}
				else
					break;
			}
		}
	}
	
	__syncthreads();

	// Write the shInd and shDist values out to global memory (TODO: coalesce the writes!).
	for( uint i = 0; i < k; i++ )
	{
		pdKNNIndices[offset + i] = shInd[threadIdx.x + i];
		pdKNNDistances[offset + i] = shDist[threadIdx.x + i];
	}

	// Write the shInd and shDist values out to global memory.
	//int index = k * blockIdx.x * blockDim.x + threadIdx.x;
	//for( uint i = 0; i < k; i++ )
	//{
	//	pdKNNIndices[index+i*THREADSPERBLOCK] = shInd[threadIdx.x+i*THREADSPERBLOCK];
	//}

	__syncthreads();
}

__global__ void KNNBruteForceCUDAKernelV2(	float3 const*	pdPosition,			// Agent positions.
											uint *			pdKNNIndices,		// Output, indices of K Nearest Neighbors in pdPosition.
											size_t const	k,					// Number of neighbors to consider.
											size_t const	numAgents			// Number of agents in the simulation.
										)
{
	int offset = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( offset >= numAgents )
		return;

	// Shared memory for local computations.
	extern __shared__ float shDist[];					// First half will be treated as the distance values.
	uint * shInd = (uint*)shDist + blockDim.x * k;		// Second half will be treated as the index values.

	// Set all elements of shDist to FLT_MAX and shInd to UINT_MAX.
	for( uint i = 0; i < k; i++ )
	{
		shDist[(threadIdx.x * k) + i] = FLT_MAX;
		shInd[(threadIdx.x * k) + i] = UINT_MAX;
	}

	// Store the positions locally.
	__shared__ float3 shPosition[THREADSPERBLOCK];
	POSITION_SH( threadIdx.x ) = POSITION( offset );

	__syncthreads();

	// For each of the agents...
	for( uint i = 0; i < numAgents; i++ )
	{
		// Test this... will likely be slower than computing k+1 and discarding the shortest...
		if( i == offset )
			continue;
		

		// Compute the distance between this agent and the one at i.
		float const dist = float3_distance( POSITION_SH( threadIdx.x ), POSITION( i ) );

		if( shDist[(threadIdx.x * k) + (k - 1)] > dist )	// Distance of the kth closest agent.
		{
			// Agent at index i is the new kth closest. Set the distance and index in shared mem.
			shDist[(threadIdx.x * k) + (k - 1)] = dist;
			shInd[(threadIdx.x * k) + (k - 1)] = i;

			// Bubble the values up...
			for( int slot = k - 2; slot >= 0; slot-- )
			{
				if( shDist[(threadIdx.x * k) + slot] > shDist[(threadIdx.x * k) + (slot + 1)] )
				{
					swap( shDist[(threadIdx.x * k) + slot], shDist[(threadIdx.x * k) + (slot + 1)] );
					swap( shInd[(threadIdx.x * k) + slot], shInd[(threadIdx.x * k) + (slot + 1)] );
				}
			}
		}
	}
	//__syncthreads();
	//// Write the shDist values out to global memory (TODO: coalesce the writes!).
	//for( uint i = 0; i < k; i++ )
	//{
	//	pdKNNIndices[offset + i] = shInd[threadIdx.x + i];
	//}
	__syncthreads();
	// This should be the coalesced version of the above...
	for( uint i = 0; i < k; i++ )
	{
		pdKNNIndices[blockIdx.x * blockDim.x + threadIdx.x * i] = shInd[threadIdx.x * i];
	}
	//__syncthreads();
}

__global__ void KNNBruteForceCUDAKernel(	float3 const*	pdPosition,			// Agent positions.
											float *			pdDistanceMatrix,	// Global storage for distance matrix.
											size_t *		pdIndexMatrix,		// The indices which match postions in pdDistanceMatrix.
											//size_t *		pdKNNIndices,		// Output, indices of K Nearest Neighbors in pdPosition.
											size_t const	k,					// Number of neighbors to consider.
											size_t const	numAgents			// Number of agents in the simulation.
										)
{
	int offset = (blockIdx.x * blockDim.x) + threadIdx.x;
	int outputOffset = offset * numAgents;

	// Check bounds.
	if( offset >= numAgents )
		return;

	// Copy the agent positions for this block to shared memory.
	__shared__ float3 shPosition[THREADSPERBLOCK];
	POSITION_SH( threadIdx.x ) = POSITION( offset );

	// For each agent in the simulation...
	for( size_t i = 0; i < numAgents; i++ )
	{
		pdDistanceMatrix[ outputOffset + i ] = float3_distance( POSITION_SH( threadIdx.x ), pdPosition[ i ] );
		pdIndexMatrix[ outputOffset + i ] = i;
	}

	__syncthreads();

	// TODO: sort pdDistanceMatrix and pdIndexMatrix
	// Currently doing externally using thrust.
	// Does it even make sense to sort them? All we want is the k lowest, surely this can be accomplished by sequentially scanning.
}