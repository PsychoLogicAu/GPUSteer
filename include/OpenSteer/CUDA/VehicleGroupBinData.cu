#include "hip/hip_runtime.h"
#include "VehicleGroupBinData.cuh"

#include "CUDAGlobals.cuh"

using namespace OpenSteer;

bin_data::bin_data( uint3 const& worldCells, float3 const& worldSize )
:	m_worldCells( worldCells ),
	m_worldSize( worldSize )
{
	// Create the cells.
	CreateCells();
}

void bin_data::CreateCells( void )
{
	float3 const step = make_float3(	m_worldSize.x / m_worldCells.x,		// width
										m_worldSize.y / m_worldCells.y,		// depth
										m_worldSize.z / m_worldCells.z );	// height

/*
Texture addressing in CUDA operates as follows. The binning representation should match it internally.
   z|
	|    y/
	|    /
	|   /
	|  /
	| /
	|/_________x
*/
	// Allocate host memory to temporarily store the 3D texture data.
	uint * phCellIndices = (uint*)malloc( m_worldCells.x * m_worldCells.y * m_worldCells.z * sizeof(uint) );

	uint offset = 0;

	for( size_t z = 0; z < m_worldCells.z; z++ )			// height - z axis
	{
		for( size_t y = 0; y < m_worldCells.y; y++ )		// depth - y axis
		{
			for( size_t x = 0; x < m_worldCells.x; x++ )	// width - x axis
			{
				// Make a bin_cell structure.
				bin_cell bc;

				//bc.iBinIndex = iBinIndex;
				bc.iCellIndex = x + (y * m_worldCells.x) + (z * m_worldCells.y * m_worldCells.x);

				// Set the offset value for the cell lookup texture.
				phCellIndices[offset++] = bc.iCellIndex;

				// TODO: set uint3 indices of m_neighborPosMin & m_neighborPosMax (?)

				// Cell is initially empty.
				bc.iBegin = 0;
				bc.iEnd = 0;
				bc.nSize = 0;

				// Set the minBounds of the cell.
				bc.minBounds.x = x * step.x;
				bc.minBounds.y = y * step.y;
				bc.minBounds.z = z * step.z;

				// Set the maxBounds of the cell.
				bc.maxBounds.x = bc.minBounds.x + step.x;
				bc.maxBounds.y = bc.minBounds.y + step.y;
				bc.maxBounds.z = bc.minBounds.z + step.z;

				m_hvCells.push_back( bc );
			}
		}
	}

	// Transfer the bin_cell structures to the device memory.
	m_dvCells = m_hvCells;

	// Prepare bin_cell index lookup texture.

	// Set the extents for the 3D texture memory.
	hipExtent const worldSize = make_hipExtent( m_worldCells.x, m_worldCells.y, m_worldCells.z );

	// Create the channel desc.
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc< uint >();

	// Allocate m_pdCellIndexArray.
	CUDA_SAFE_CALL( hipMalloc3DArray( &m_pdCellIndexArray, &channelDesc, worldSize ) );

	// Copy data to 3D array.
	hipMemcpy3DParms copyParms = {0};
	copyParms.srcPtr = make_hipPitchedPtr( (void*)phCellIndices, worldSize.width*sizeof(uint), worldSize.width, worldSize.height );
	copyParms.dstArray = m_pdCellIndexArray;
	copyParms.extent = worldSize;
	copyParms.kind = hipMemcpyHostToDevice;
	CUDA_SAFE_CALL( hipMemcpy3D( &copyParms ) );

	// Free dynamic memory.
	free( phCellIndices );
}