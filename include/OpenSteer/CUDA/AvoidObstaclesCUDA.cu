#include "hip/hip_runtime.h"
#include "AvoidObstaclesCUDA.cuh"

using namespace OpenSteer;

// Kernel function prototype.
extern "C"
{
	__global__ void SteerToAvoidObstaclesKernel(	uint const*		pdKNNIndices,			// In:	Indices of the K Nearest Obstacles.
													float const*	pdKNNDistances,			// In:	Distances to the K Nearest Obstacles.
													size_t const	k,
												
													float3 const*	pdPosition,				// In:	Agent positions.
													float3 const*	pdDirection,			// In:	Agent directions.
													float3 const*	pdSide,
													float3 const*	pdUp,
													float const*	pdRadius,				// In:	Agent radii.
													float const*	pdSpeed,				// In:	Agent speeds.

													float3 const*	pdObstaclePosition,		// In:	Obstacle positions.
													float const*	pdObstacleRadius,		// In:	Obstacle radii.

													float const		minTimeToCollision,
		
													float3 *		pdSteering,				// Out:	Agent steering vectors.
													
													uint const		numAgents,				// In:	Number of agents.
													uint const		numObstacles,			// In:	Number of obstacles.
													float const		fWeight					// In:	Weight for this kernel
													);
}


AvoidObstaclesCUDA::AvoidObstaclesCUDA( AgentGroup * pAgentGroup, ObstacleGroup * pObstacleGroup, KNNData * pKNNData, float const fMinTimeToCollision, float const fWeight )
:	AbstractCUDAKernel( pAgentGroup, fWeight ),
	m_pObstacleGroup( pObstacleGroup ),
	m_fMinTimeToCollision( fMinTimeToCollision ),
	m_pKNNData( pKNNData )
{
	// Nothing to do.
}

void AvoidObstaclesCUDA::init(void)
{

}

void AvoidObstaclesCUDA::run(void)
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	uint const*		pdKNNIndices		= m_pKNNData->pdKNNIndices();
	float const*	pdKNNDistances		= m_pKNNData->pdKNNDistances();
	uint const&		k					= m_pKNNData->k();

	float3 const*	pdPosition			= m_pAgentGroupData->pdPosition();
	float3 const*	pdDirection			= m_pAgentGroupData->pdDirection();
	float3 const*	pdSide				= m_pAgentGroupData->pdSide();
	float3 const*	pdUp				= m_pAgentGroupData->pdUp();
	float const*	pdRadius			= m_pAgentGroupConst->pdRadius();
	float const*	pdSpeed				= m_pAgentGroupData->pdSpeed();

	float3 const*	pdObstaclePosition	= m_pObstacleGroup->pdPosition();
	float const*	pdObstacleRadius	= m_pObstacleGroup->pdRadius();

	float3 *		pdSteering			= m_pAgentGroupData->pdSteering();
	
	uint const&		numAgents			= m_pAgentGroup->Size();
	uint const&		numObstacles		= m_pObstacleGroup->Size();

	size_t shMemSize = k * THREADSPERBLOCK * (sizeof(uint) + sizeof(float));

	SteerToAvoidObstaclesKernel<<< grid, block, shMemSize >>>(	pdKNNIndices, pdKNNDistances, k,
																pdPosition, pdDirection, pdSide, pdUp, pdRadius, pdSpeed,
																pdObstaclePosition, pdObstacleRadius,
																m_fMinTimeToCollision, 
																pdSteering, 
																numAgents, numObstacles,
																m_fWeight
																);
	cutilCheckMsg( "AvoidObstaclesCUDAKernel failed." );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

void AvoidObstaclesCUDA::close(void)
{
	// Device data has changed. Instruct the AgentGroup it needs to synchronize the host.
	m_pAgentGroup->SetSyncHost();
}
