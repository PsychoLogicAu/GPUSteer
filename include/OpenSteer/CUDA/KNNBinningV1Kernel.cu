#include "hip/hip_runtime.h"
#include "KNNBinningV1.cuh"

#include "../VectorUtils.cuh"

#include "CUDAKernelGlobals.cuh"

using namespace OpenSteer;

// Texture references.
texture< uint, hipTextureType3D, hipReadModeElementType >		texCellIndicesNormalized;
texture< uint, hipTextureType3D, hipReadModeElementType >		texCellIndices;
texture< float4, hipTextureType1D, hipReadModeElementType >	texPosition;

__constant__ float3		constWorldSizeV1;
__constant__ float3		constWorldStepV1;
__constant__ float3		constWorldStepNormalizedV1;
__constant__ uint3		constWorldCellsV1;

#define CELL_INDEX( pos )	( tex3D( texCellIndices, pos.x, pos.y, pos.z ) )	// <--- for choke point.
//#define CELL_INDEX( pos )	( tex3D( texCellIndices, pos.x, pos.z, pos.y ) )	// <--- for boids.

// Kernel declarations.
extern "C"
{
	// Bind the textures to the input hipArray.
	__host__ void KNNBinningV1BindTexture(			hipArray * pCudaArray );
	// Unbind the textures.
	__host__ void KNNBinningV1UnbindTexture( void );

	__host__ void KNNBinningV1KernelBindTextures(			uint const*		pdBCellStart,
														uint const*		pdBCellEnd,
														uint const*		pdBIndices,
														float4 const*	pdBPositionSorted,
														uint const		numCells,
														uint const		numB
														);

	__host__ void KNNBinningV1KernelUnbindTextures( void );
	__host__ void KNNBinningV1ReorderDBBindTextures(		float4 const*	pdPosition,
														uint const		numAgents
														);
	__host__ void KNNBinningV1ReorderDBUnbindTextures( void );

	// Kernel to set initial bin indices of vehicles in the simulation.
	__global__ void KNNBinningV1BuildDB(					float4 const*	pdPosition,				// In:	Positions of each agent.
														size_t *		pdAgentIndices,			// Out:	Indices of each agent.
														size_t *		pdCellIndices,			// Out:	Indices of the cell each agent is in.
														size_t const	numAgents
														);

	// Reorder the positions on pdCellIndices, and compute the cell start and end indices.
	__global__ void KNNBinningV1ReorderDB(				uint const*		pdAgentIndices,		// In: (sorted) agent index.
														uint const*		pdCellIndices,		// In: (sorted) cell index agent is in.

														float4 *		pdPositionSorted,	// Out: Sorted agent positions.

														uint *			pdCellStart,		// Out: Start index of this cell in pdCellIndices.
														uint *			pdCellEnd,			// Out: End index of this cell in pdCellIndices.

														size_t const	numAgents
														);

	__global__ void KNNBinningV1Kernel(					// Group A
														float4 const*	pdAPositionSorted,			// In:	Sorted group A positions.
														uint const*		pdAIndices,					// In:	Sorted group A indices
														uint const		numA,						// In:	Size of group A.

														// Cell neighbor info.
														int const		radius,						// In:	Search radius (in cells) to consider.

														// Output data.
														uint *			pdKNNIndices,				// Out:	Indices of K Nearest Neighbors in pdPosition.
														float *			pdKNNDistances,				// Out:	Distances of the K Nearest Neighbors in pdPosition.
														uint const		k,							// In:	Number of neighbors to consider.

														uint const		numB,						// In:	Size of group B.
														bool const		groupWithSelf				// In:	Are we testing this group with itself? (group A == group B)
														);
}

__inline__ __device__ int3 ComputeCellPos( volatile float3 const worldPosition )
{
	int3 cellPos;

	cellPos.x = (worldPosition.x + 0.5f * constWorldSizeV1.x) / constWorldStepV1.x;
	cellPos.y = (worldPosition.y + 0.5f * constWorldSizeV1.y) / constWorldStepV1.y;
	cellPos.z = (worldPosition.z + 0.5f * constWorldSizeV1.z) / constWorldStepV1.z;

	return cellPos;
}

__host__ void KNNBinningV1BindTexture( hipArray * pdCudaArray )
{
	static hipChannelFormatDesc const channelDesc = hipCreateChannelDesc< uint >();

	texCellIndices.normalized = false;
	texCellIndices.filterMode = hipFilterModePoint;
	// Wrap out of bounds coordinates.
	texCellIndices.addressMode[0] = hipAddressModeWrap;
	texCellIndices.addressMode[1] = hipAddressModeWrap;
	texCellIndices.addressMode[2] = hipAddressModeWrap;

	CUDA_SAFE_CALL( hipBindTextureToArray( texCellIndices, pdCudaArray, channelDesc ) );
}

__host__ void KNNBinningV1UnbindTexture( void )
{
	//CUDA_SAFE_CALL( hipUnbindTexture( texCellIndicesNormalized ) );

	CUDA_SAFE_CALL( hipUnbindTexture( texCellIndices ) );
}

__global__ void KNNBinningV1BuildDB(	float4 const*	pdPosition,				// In:	Positions of each agent.
									size_t *		pdAgentIndices,			// Out:	Indices of each agent.
									size_t *		pdCellIndices,			// Out:	Indices of the cell each agent is in.
									size_t const	numAgents
									)
{
	// Offset of this agent in the global array.
	int const index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( index >= numAgents )
		return;

	// Copy the positions to shared memory.
	__shared__ float3 shPosition[THREADSPERBLOCK];
	POSITION_SH( threadIdx.x ) = POSITION_F3( index );

	// Write the agent's cell index out to global memory.
	pdCellIndices[index] = CELL_INDEX( ComputeCellPos( POSITION_SH( threadIdx.x ) ) );

	// Write the agent's index out to global memory.
	pdAgentIndices[index] = index;
}

__host__ void KNNBinningV1ReorderDBBindTextures(	float4 const*	pdPosition,
												uint const		numAgents
												)
{
	static hipChannelFormatDesc const float4ChannelDesc = hipCreateChannelDesc< float4 >();

	CUDA_SAFE_CALL( hipBindTexture( NULL, texPosition, pdPosition, float4ChannelDesc, numAgents * sizeof(float4) ) );
}


__host__ void KNNBinningV1ReorderDBUnbindTextures( void )
{
	CUDA_SAFE_CALL( hipUnbindTexture( texPosition ) );
}

__global__ void KNNBinningV1ReorderDB(	uint const*		pdAgentIndices,		// In: (sorted) agent index.
										uint const*		pdCellIndices,		// In: (sorted) cell index agent is in.

										float4 *		pdPositionSorted,	// Out: Sorted agent positions.

										uint *			pdCellStart,		// Out: Start index of this cell in pdCellIndices.
										uint *			pdCellEnd,			// Out: End index of this cell in pdCellIndices.

										size_t const	numAgents
										)
{
	// Offset of this agent.
	int const index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( index >= numAgents )
		return;

	__shared__ uint shCellIndices[THREADSPERBLOCK+1];

	// Shared memory so we can coalesce the writes of sorted data to global memory.
	__shared__ float4 shPositionSorted[THREADSPERBLOCK];

	// Read the cell index of this agent.
	uint iCellIndex = pdCellIndices[ index ];
	__syncthreads();
	
	// Store cell index data in shared memory so that we can look 
	// at the neighboring agent's value without two reads per thread.
	shCellIndices[ threadIdx.x + 1 ] = iCellIndex;

	if( index > 0 && threadIdx.x == 0 )
	{
		// First thread in block must load neighbor agent cell index.
		shCellIndices[0] = pdCellIndices[ index - 1 ];
	}

	__syncthreads();

	// If this agent has a different cell index to the previous
	// agent then it must be the first in the cell,
	// so store the index of this agent in the cell.
	// As it isn't the first agent, it must also be the cell end of
	// the previous particle's cell

	if( index == 0 || iCellIndex != shCellIndices[ threadIdx.x ] )
	{
		pdCellStart[ iCellIndex ] = index;
		if( index > 0 )
			pdCellEnd[ shCellIndices[ threadIdx.x ] ] = index;
	}

	// If this is the last agent, the end index for the cell will be index + 1
	if( index == (numAgents - 1) )
	{
		pdCellEnd[ iCellIndex ] = index + 1;
	}

	// Use the sorted index to reorder the position/direction/speed data.
	uint const iSortedIndex = pdAgentIndices[ index ];

	shPositionSorted[ threadIdx.x ] = tex1Dfetch( texPosition, iSortedIndex );

	// Write to global memory.
	pdPositionSorted[ index ] = shPositionSorted[ threadIdx.x ];
}

// Textures used by KNNBinningKernel.
texture< uint, hipTextureType1D, hipReadModeElementType>		texBCellStart;
texture< uint, hipTextureType1D, hipReadModeElementType>		texBCellEnd;
texture< uint, hipTextureType1D, hipReadModeElementType>		texBIndices;
texture< float4, hipTextureType1D, hipReadModeElementType>	texBPositionSorted;

__host__ void KNNBinningV1KernelBindTextures(	uint const*		pdBCellStart,
											uint const*		pdBCellEnd,
											uint const*		pdBIndices,
											float4 const*	pdBPositionSorted,
											uint const		numCells,
											uint const		numB
											)
{
	static hipChannelFormatDesc const uintChannelDesc = hipCreateChannelDesc< uint >();
	static hipChannelFormatDesc const float4ChannelDesc = hipCreateChannelDesc< float4 >();

	CUDA_SAFE_CALL( hipBindTexture( NULL, texBCellStart, pdBCellStart, uintChannelDesc, numCells * sizeof(uint) ) );
	CUDA_SAFE_CALL( hipBindTexture( NULL, texBCellEnd, pdBCellEnd, uintChannelDesc, numCells * sizeof(uint) ) );
	CUDA_SAFE_CALL( hipBindTexture( NULL, texBIndices, pdBIndices, uintChannelDesc, numB * sizeof(uint) ) );
	CUDA_SAFE_CALL( hipBindTexture( NULL, texBPositionSorted, pdBPositionSorted, float4ChannelDesc, numB * sizeof(float4) ) );
}

__host__ void KNNBinningV1KernelUnbindTextures( void )
{
	CUDA_SAFE_CALL( hipUnbindTexture( texBCellStart ) );
	CUDA_SAFE_CALL( hipUnbindTexture( texBCellEnd ) );
	CUDA_SAFE_CALL( hipUnbindTexture( texBIndices ) );
	CUDA_SAFE_CALL( hipUnbindTexture( texBPositionSorted ) );
}

__global__ void KNNBinningV1Kernel(	// Group A
									float4 const*	pdAPositionSorted,			// In:	Sorted group A positions.
									uint const*		pdAIndices,					// In:	Sorted group A indices
									uint const		numA,						// In:	Size of group A.

									// Cell neighbor info.
									int const		radius,						// In:	Search radius (in cells) to consider.

									// Output data.
									uint *			pdKNNIndices,				// Out:	Indices of K Nearest Neighbors in pdPosition.
									float *			pdKNNDistances,				// Out:	Distances of the K Nearest Neighbors in pdPosition.
									uint const		k,							// In:	Number of neighbors to consider.

									uint const		numB,						// In:	Size of group B.
									bool const		groupWithSelf				// In:	Are we testing this group with itself? (group A == group B)
									)
{
	// Index of this agent.
	int const AIndexSorted = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Check bounds.
	if( AIndexSorted >= numA )
		return;

	__shared__ float3 shAPosition[THREADSPERBLOCK];

	// Shared memory for local priority queue computations.
	extern __shared__ uint shKNNIndices[];
	float * shKNNDistances = (float*)shKNNIndices + THREADSPERBLOCK * k;

	// Set all elements of shKNNDistances to FLT_MAX, shKNNIndices to UINT_MAX.
	for( uint i = 0; i < k; i++ )
	{
		shKNNIndices[(threadIdx.x * k) + i] = UINT_MAX;
		shKNNDistances[(threadIdx.x * k) + i] = FLT_MAX;
	}

	// Store this thread's index and cell index in registers.
	uint const		AIndex					= pdAIndices[ AIndexSorted ];

	// Coalesce read the positions.
	shAPosition[ threadIdx.x ] = make_float3( pdAPositionSorted[ AIndexSorted ] );

	int3 const cellPos = ComputeCellPos( shAPosition[ threadIdx.x ] );

	for( int dy = -radius; dy <= radius; dy++ )				// World height.
		for( int dz = -radius; dz <= radius; dz++ )			// World depth.
			for( int dx = -radius; dx <= radius; dx++ )		// World width.
			{
				int3 neighborPos = make_int3( cellPos.x + dx, cellPos.y + dy, cellPos.z + dz );
				uint const cellIndex = CELL_INDEX( neighborPos );

				// For each member of group B in the cell...
				for( uint BIndexSorted = tex1Dfetch( texBCellStart, cellIndex ) /*pdBCellStart[ cellIndex ]*/; BIndexSorted < tex1Dfetch( texBCellEnd, cellIndex ) /*pdBCellEnd[ cellIndex ]*/; BIndexSorted++ )
				{
					// Get the index of the other agent (unsorted).
					uint const BIndex = tex1Dfetch( texBIndices, BIndexSorted ) /*pdBIndices[ BIndexSorted ]*/;

					// Do not include self.
					if( groupWithSelf && AIndex == BIndex )
						continue;

					// Compute the distance between this thread'a A position and the B position at otherIndexSorted
					float const dist = float3_distance( shAPosition[threadIdx.x], make_float3( tex1Dfetch( texBPositionSorted, BIndexSorted ) ) );

					if( dist < shKNNDistances[(threadIdx.x * k) + (k - 1)] )	// Distance of the kth closest agent.
					{
						// Agent at index BIndex is the new (at least) kth closest. Set the distance and index in shared mem.
						shKNNDistances[(threadIdx.x * k) + (k - 1)] = dist;
						shKNNIndices[(threadIdx.x * k) + (k - 1)] = BIndex;

						// Bubble the values up...
						for( int slot = k - 2; slot >= 0; slot-- )
						{
							if( shKNNDistances[(threadIdx.x * k) + slot] > shKNNDistances[(threadIdx.x * k) + (slot + 1)] )
							{
								swap( shKNNDistances[(threadIdx.x * k) + slot], shKNNDistances[(threadIdx.x * k) + (slot + 1)] );
								swap( shKNNIndices[(threadIdx.x * k) + slot], shKNNIndices[(threadIdx.x * k) + (slot + 1)] );
							}
							else
								break;
						}
					}
				}
			}

	__syncthreads();

	// Write the shKNNIndices and shKNNDistances values out to global memory.
	for( uint i = 0; i < k; i++ )
	{
		pdKNNIndices[AIndex*k + i] = shKNNIndices[threadIdx.x*k + i];
		pdKNNDistances[AIndex*k + i] = shKNNDistances[threadIdx.x*k + i];
	}
	__syncthreads();
}