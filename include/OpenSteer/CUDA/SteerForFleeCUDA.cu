#include "hip/hip_runtime.h"
#include "SteerForFleeCUDA.h"

#include "../VehicleGroupData.cuh"

using namespace OpenSteer;

// Kernel function prototype.
extern "C"
{
	__global__ void SteerForFleeCUDAKernel(	float3 const* pdPosition, float3 const* pdForward, float3 * pdSteering,
											float3 const target, size_t const numAgents );
}

SteerForFleeCUDA::SteerForFleeCUDA( VehicleGroup * pVehicleGroup, const float3 &target )
:	AbstractCUDAKernel( pVehicleGroup ),
	m_target( target )
{ }

void SteerForFleeCUDA::init(void)
{ }

void SteerForFleeCUDA::run(void)
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	// Gather required device pointers.
	float3 const* pdPosition = m_pVehicleGroupData->pdPosition();
	float3 const* pdForward = m_pVehicleGroupData->pdForward();
	float3 * pdSteering = m_pVehicleGroupData->pdSteering();

	SteerForFleeCUDAKernel<<< grid, block >>>( pdPosition, pdForward, pdSteering, m_target, getNumAgents() );
	cutilCheckMsg( "SteerForFleeCUDAKernel failed." );

	hipDeviceSynchronize();
}

void SteerForFleeCUDA::close(void)
{
	// Device data has changed. Instruct the VehicleGroup it needs to synchronize the host.
	m_pVehicleGroup->SetSyncHost();
}
