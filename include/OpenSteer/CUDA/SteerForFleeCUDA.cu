#include "hip/hip_runtime.h"
#include "SteerForFleeCUDA.h"

#include "../VehicleData.cu"

using namespace OpenSteer;

// Kernel function prototype.
extern "C"
{
	__global__ void SteerForFleeCUDAKernel(vehicle_data *vehicleData, float3 target, int numAgents);
}

SteerForFleeCUDA::SteerForFleeCUDA(VehicleGroup *vehicleGroup, const float3 &target)
: AbstractCUDAKernel(vehicleGroup)
{
	m_target = target;
}

void SteerForFleeCUDA::init(void)
{
	// Allocate device memory.
	HANDLE_ERROR(hipMalloc((void**)&m_pdVehicleData, getDataSizeInBytes()));

	// Copy data to device memory.
	HANDLE_ERROR(hipMemcpy(m_pdVehicleData, (void*)getVehicleData(), getDataSizeInBytes(), hipMemcpyHostToDevice));
}

void SteerForFleeCUDA::run(void)
{
	dim3 grid = gridDim();
	dim3 block = blockDim();

	SteerForFleeCUDAKernel<<<grid, block>>>(m_pdVehicleData, m_target, getNumberOfAgents());

	hipDeviceSynchronize();
}

void SteerForFleeCUDA::close(void)
{
	// Copy vehicle data back to the host memory.
	HANDLE_ERROR(hipMemcpy((void*)getVehicleData(), m_pdVehicleData, getDataSizeInBytes(), hipMemcpyDeviceToHost));

	// Deallocate device memory
	HANDLE_ERROR(hipFree(m_pdVehicleData));
	m_pdVehicleData = NULL;
}
