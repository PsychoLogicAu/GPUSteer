#include "hip/hip_runtime.h"
#include "SteerForPursuitCUDA.h"

#include "../VehicleGroupData.cuh"
#include "../VectorUtils.cuh"

#include "CUDAKernelGlobals.cuh"

using namespace OpenSteer;

extern "C"
{
	__global__ void SteerForPursuitCUDAKernel(	float3 * pdSteering, float3 const* pdPosition, float3 const* pdForward, float const* pdSpeed, 
												float3 const targetPosition, float3 const targetForward, float3 const targetVelocity, float const targetSpeed,
												size_t const numAgents, float const maxPredictionTime )
	{
		int offset = (blockIdx.x * blockDim.x) + threadIdx.x;

		// Check bounds.
		if( offset >= numAgents )
			return;

		// Declare shared memory.
		__shared__ float3 shSteering[THREADSPERBLOCK];
		__shared__ float3 shPosition[THREADSPERBLOCK];
		__shared__ float3 shForward[THREADSPERBLOCK];
		__shared__ float shSpeed[THREADSPERBLOCK];

		FLOAT3_GLOBAL_READ( shSteering, pdSteering );
		FLOAT3_GLOBAL_READ( shPosition, pdPosition );
		FLOAT3_GLOBAL_READ( shForward, pdForward );
		
		SPEED_SH( threadIdx.x ) = SPEED( offset );
		__syncthreads();

		// If we already have a steering vector set, do nothing.
		if( ! float3_equals( STEERING_SH( threadIdx.x ), float3_zero() ) )
			return;

		// If the target is ahead, just seek to its current position.
		float3 toTarget = float3_subtract( targetPosition, POSITION_SH( threadIdx.x ) );
		float relativeHeading = float3_dot( FORWARD_SH( threadIdx.x ), targetForward );

		if( float3_dot( toTarget, FORWARD_SH( threadIdx.x ) ) > 0 && (relativeHeading < -0.95f))
		{
			// Get the desired velocity.
			float3 desiredVelocity = float3_subtract( targetPosition, POSITION_SH( threadIdx.x ) );

			// Set the steering vector.
			STEERING_SH( threadIdx.x ) = float3_subtract( desiredVelocity, FORWARD_SH( threadIdx.x ) );
		}
		else
		{
			float lookAheadTime = float3_length( toTarget ) / ( SPEED_SH( threadIdx.x ) + targetSpeed );
			float3 newTarget = float3_add( targetPosition, float3_scalar_multiply( targetVelocity, (maxPredictionTime < lookAheadTime) ? maxPredictionTime : lookAheadTime ) );

			// Get the desired velocity.
			float3 desiredVelocity = float3_subtract( newTarget, POSITION_SH( threadIdx.x ) );

			// Set the steering vector.
			STEERING_SH( threadIdx.x ) = float3_subtract( desiredVelocity, FORWARD_SH( threadIdx.x ) );
		}

		__syncthreads();

		FLOAT3_GLOBAL_WRITE( pdSteering, shSteering );
	}
}
